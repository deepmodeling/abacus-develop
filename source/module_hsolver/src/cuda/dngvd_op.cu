#include "hip/hip_runtime.h"
#include "module_hsolver/include/dngvd_op.h"


#include <hipsolver.h>


namespace hsolver {

template <>
void dngvd_op<double, psi::DEVICE_GPU>::operator()(
        const psi::DEVICE_GPU* d,
        const int row,
        const int col,
        const std::complex<double>* A,
        const std::complex<double>* B,
        const int m,
        double* W, 
        std::complex<double>* V)
{
    createBLAShandle();
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, A, A);
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, B, B);
    // init A_eigenvectors and all_W 
    double2* A_eigenvectors;
    hipMalloc ((void**)&A_eigenvectors, sizeof(double2) * row * col);
    hipMemcpy(A_eigenvectors, A, sizeof(double2) * row * col, hipMemcpyDeviceToDevice);
    double* all_W ;
    hipMalloc ((void**)&all_W, sizeof(double) * row);

    // init
    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);
    int * devInfo ;
    hipMalloc ((void**)&devInfo, sizeof(int));
    
    // calculate the sizes needed for pre-allocated buffer.
    int lwork = 0;
    hipsolverDnZhegvd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1,        // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR,   // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_LOWER,
        row,
        A_eigenvectors,
        col,
        (double2*)B,
        col,
        all_W,
        &lwork);

    // allocate memery
    hipDoubleComplex *d_work;
    hipMalloc((void**)&d_work, sizeof(hipDoubleComplex)*lwork);

    // compute eigenvalues and eigenvectors. 
    hipsolverDnZhegvd(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1,        // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR,   // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_LOWER,
        row,
        A_eigenvectors,
        col,
        (double2*)B,
        col,
        all_W,
        d_work,
        lwork,
        devInfo);

    hipDeviceSynchronize();

    // get eigenvalues and eigenvectors.  only m !
    hipMemcpy(W, all_W, sizeof(double)*m, hipMemcpyDeviceToDevice);
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, A_eigenvectors, A_eigenvectors);
    hipMemcpy(V, A_eigenvectors, sizeof(std::complex<double>)*col*m, hipMemcpyDeviceToDevice);
    // free the buffer
    hipFree(d_work);
    // free resources and destroy
    hipFree(A_eigenvectors);
    hipFree(all_W);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH); 
    destoryBLAShandle();
}


}