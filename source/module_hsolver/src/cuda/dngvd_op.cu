#include "hip/hip_runtime.h"
#include "module_hsolver/include/dngvd_op.h"


#include <hipsolver.h>


namespace hsolver {

template <>
void dngvd_op<double, psi::DEVICE_GPU>::operator()(
        const psi::DEVICE_GPU* d,
        const int row,
        const int col,
        const std::complex<double>* A,
        const std::complex<double>* B,
        const int m,
        double* W, 
        std::complex<double>* V)
{
    createBLAShandle();
    // init A_eigenvectors, transpose_B and all_W 
    double2 *A_eigenvectors, *transpose_B;
    hipMalloc ((void**)&A_eigenvectors, sizeof(double2) * row * col);
    hipMalloc ((void**)&transpose_B, sizeof(double2) * row * col);
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, A, (std::complex<double>*)A_eigenvectors);
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, B, (std::complex<double>*)transpose_B);
    double* all_W ;
    hipMalloc ((void**)&all_W, sizeof(double) * row);

    // init
    hipsolverHandle_t cusolverH;
    hipsolverDnCreate(&cusolverH);
    int * devInfo ;
    hipMalloc ((void**)&devInfo, sizeof(int));
    
    // calculate the sizes needed for pre-allocated buffer.
    int lwork = 0;
    hipsolverDnZhegvd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1,        // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR,   // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_LOWER,
        row,
        A_eigenvectors,
        col,
        transpose_B,
        col,
        all_W,
        &lwork);

    // allocate memery
    hipDoubleComplex *d_work;
    hipMalloc((void**)&d_work, sizeof(hipDoubleComplex)*lwork);

    // compute eigenvalues and eigenvectors. 
    hipsolverDnZhegvd(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1,        // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR,   // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_LOWER,
        row,
        A_eigenvectors,
        col,
        transpose_B,
        col,
        all_W,
        d_work,
        lwork,
        devInfo);

    hipDeviceSynchronize();

    // get eigenvalues and eigenvectors.  only m !
    hipMemcpy(W, all_W, sizeof(double)*m, hipMemcpyDeviceToDevice);
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, (std::complex<double>*)A_eigenvectors, (std::complex<double>*)A_eigenvectors);
    hipMemcpy(V, A_eigenvectors, sizeof(std::complex<double>)*col*m, hipMemcpyDeviceToDevice);
    // free the buffer
    hipFree(d_work);
    // free resources and destroy
    hipFree(A_eigenvectors);
    hipFree(all_W);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH); 
    destoryBLAShandle();
}


}