#include "hip/hip_runtime.h"
#include "module_hsolver/include/dngvd_op.h"
#include "module_hsolver/include/math_kernel.h"

#include <hipsolver.h>


namespace hsolver {

template <>
void dngvd_op<double, psi::DEVICE_GPU>::operator()(
        const psi::DEVICE_GPU* d,
        const int row,
        const int col,
        const std::complex<double>* A,
        const std::complex<double>* B,
        const int m,
        double* W, 
        std::complex<double>* V)
{
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, A, A);
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, B, B);
    // init A_eigenvectors and all_W 
    double2* A_eigenvectors;
    checkCudaErrors( hipMalloc ((void**)&A_eigenvectors, sizeof(double2) * row * col) );
    checkCudaErrors( hipMemcpy(A_eigenvectors, A, sizeof(double2) * row * col, hipMemcpyDeviceToDevice) );
    double* all_W ;
    checkCudaErrors( hipMalloc ((void**)&all_W, sizeof(double) * row) );

    // init
    hipsolverHandle_t cusolverH;
    checkCudaErrors( hipsolverDnCreate(&cusolverH) );
    int * devInfo ;
    checkCudaErrors( hipMalloc ((void**)&devInfo, sizeof(int)) );
    
    // calculate the sizes needed for pre-allocated buffer.
    int lwork = 0;
    checkCudaErrors( 
        hipsolverDnZhegvd_bufferSize(
            cusolverH,
            HIPSOLVER_EIG_TYPE_1,        // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
            HIPSOLVER_EIG_MODE_VECTOR,   // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
            HIPBLAS_FILL_MODE_LOWER,
            row,
            A_eigenvectors,
            col,
            (double2*)B,
            col,
            all_W,
            &lwork)
    );

    // allocate memery
    hipDoubleComplex *d_work;
    checkCudaErrors( hipMalloc((void**)&d_work, sizeof(hipDoubleComplex)*lwork) );

    // compute eigenvalues and eigenvectors. 
    checkCudaErrors(
        hipsolverDnZhegvd(
            cusolverH,
            HIPSOLVER_EIG_TYPE_1,        // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
            HIPSOLVER_EIG_MODE_VECTOR,   // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
            HIPBLAS_FILL_MODE_LOWER,
            row,
            A_eigenvectors,
            col,
            (double2*)B,
            col,
            all_W,
            d_work,
            lwork,
            devInfo)
    );

    checkCudaErrors( hipDeviceSynchronize() );

    // get eigenvalues and eigenvectors.  only m !
    checkCudaErrors( hipMemcpy(W, all_W, sizeof(double)*m, hipMemcpyDeviceToDevice) );
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, A_eigenvectors, A_eigenvectors);
    checkCudaErrors( hipMemcpy(V, A_eigenvectors, sizeof(std::complex<double>)*col*m, hipMemcpyDeviceToDevice) );
    // free the buffer
    checkCudaErrors( hipFree(d_work) );
    // free resources and destroy
    checkCudaErrors( hipFree(A_eigenvectors) );
    checkCudaErrors( hipFree(all_W) );
    checkCudaErrors( hipFree(devInfo) );
    checkCudaErrors( hipsolverDnDestroy(cusolverH) ); 
}


}