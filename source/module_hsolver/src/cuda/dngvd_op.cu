#include "hip/hip_runtime.h"
#include "module_hsolver/include/dngvd_op.h"
#include "src_pdiag/hip/hip_runtime_api.h"

#include <hipsolver.h>

#define cusolverErrcheck(res)                      \
    {                                              \
        cusolverAssert((res), __FILE__, __LINE__); \
    }

// cuSOLVER API errors
static const char* _cusolverGetErrorEnum(hipsolverStatus_t error)
{
    switch (error)
    {
    case HIPSOLVER_STATUS_SUCCESS:
        return "HIPSOLVER_STATUS_SUCCESS";
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
        return "HIPSOLVER_STATUS_NOT_INITIALIZED";
    case HIPSOLVER_STATUS_ALLOC_FAILED:
        return "HIPSOLVER_STATUS_ALLOC_FAILED";
    case HIPSOLVER_STATUS_INVALID_VALUE:
        return "HIPSOLVER_STATUS_INVALID_VALUE";
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
        return "HIPSOLVER_STATUS_ARCH_MISMATCH";
    case HIPSOLVER_STATUS_MAPPING_ERROR:
        return "HIPSOLVER_STATUS_MAPPING_ERROR";
    case HIPSOLVER_STATUS_EXECUTION_FAILED:
        return "HIPSOLVER_STATUS_EXECUTION_FAILED";
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
        return "HIPSOLVER_STATUS_INTERNAL_ERROR";
    case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSOLVER_STATUS_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_NOT_SUPPORTED ";
    case HIPSOLVER_STATUS_ZERO_PIVOT:
        return "HIPSOLVER_STATUS_ZERO_PIVOT";
    case CUSOLVER_STATUS_INVALID_LICENSE:
        return "CUSOLVER_STATUS_INVALID_LICENSE";
    }
    return "<unknown>";
}

inline void cusolverAssert(hipsolverStatus_t code, const char* file, int line, bool abort = true)
{
    if (code != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuSOLVER Assert: %s %s %d\n", _cusolverGetErrorEnum(code), file, line);
        if (abort)
            exit(code);
    }
}

namespace hsolver
{

static hipsolverHandle_t cusolver_H = nullptr;

void createCUSOLVERhandle()
{
    if (cusolver_H == nullptr)
    {
        cusolverErrcheck(hipsolverDnCreate(&cusolver_H));
    }
}

void destoryCUSOLVERhandle()
{
    if (cusolver_H != nullptr)
    {
        cusolverErrcheck(hipsolverDnDestroy(cusolver_H));
        cusolver_H = nullptr;
    }
}

static inline
void xhegvd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<float> * A, const int& lda,
        std::complex<float> * B, const int& ldb,
        float * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    float2 * work = nullptr;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnChegvd_bufferSize(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const float2 *>(A), lda,
                                                 reinterpret_cast<const float2 *>(B), ldb, W, &lwork));
    // allocate memery
    checkCudaErrors(hipMalloc((void**)&work, sizeof(float2) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnChegvd(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                      reinterpret_cast<float2 *>(A), lda, reinterpret_cast<float2 *>(B), ldb, W, work, lwork, devInfo));

    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    // free the buffer
    checkCudaErrors(hipFree(work));
    checkCudaErrors(hipFree(devInfo));
}

static inline
void xhegvd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<double> * A, const int& lda,
        std::complex<double> * B, const int& ldb,
        double * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    double2 * work = nullptr;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnZhegvd_bufferSize(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const double2 *>(A), lda,
                                                 reinterpret_cast<const double2 *>(B), ldb, W, &lwork));
    // allocate memery
    checkCudaErrors(hipMalloc((void**)&work, sizeof(double2) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZhegvd(cusolver_H, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                      reinterpret_cast<double2 *>(A), lda, reinterpret_cast<double2 *>(B), ldb, W, work, lwork, devInfo));

    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    // free the buffer
    checkCudaErrors(hipFree(work));
    checkCudaErrors(hipFree(devInfo));
}

static inline
void xheevd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<float> * A, const int& lda,
        float * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    float2 * work = nullptr;
    hipsolverHandle_t cusolverH = {};
    cusolverErrcheck(hipsolverDnCreate(&cusolverH));
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnCheevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const float2 *>(A), lda, W, &lwork));
    // allocate memery
    checkCudaErrors(hipMalloc((void**)&work, sizeof(float2) * lwork));
    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnCheevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, uplo, n, reinterpret_cast<float2 *>(A), lda, W, work, lwork, devInfo));

    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    checkCudaErrors(hipFree(work));
    checkCudaErrors(hipFree(devInfo));
    cusolverErrcheck(hipsolverDnDestroy(cusolverH));
}

static inline
void xheevd_wrapper (
        const hipblasFillMode_t& uplo,
        const int& n,
        std::complex<double> * A, const int& lda,
        double * W)
{
    // prepare some values for hipsolverDnZhegvd_bufferSize
    int * devInfo = nullptr;
    int lwork = 0, info_gpu = 0;
    double2 * work = nullptr;
    hipsolverHandle_t cusolverH = {};
    cusolverErrcheck(hipsolverDnCreate(&cusolverH));
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    cusolverErrcheck(hipsolverDnZheevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                                 reinterpret_cast<const double2 *>(A), lda, W, &lwork));
    // allocate memery
    checkCudaErrors(hipMalloc((void**)&work, sizeof(double2) * lwork));
    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZheevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, uplo, n,
                                      reinterpret_cast<double2 *>(A), lda, W, work, lwork, devInfo));

    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);
    checkCudaErrors(hipFree(work));
    checkCudaErrors(hipFree(devInfo));
    cusolverErrcheck(hipsolverDnDestroy(cusolverH));
}

template <typename FPTYPE>
struct dngvx_op<FPTYPE, psi::DEVICE_GPU> {
    void operator()(
            const psi::DEVICE_GPU * d,
            const int nstart,
            const int ldh,
            const std::complex<FPTYPE> *A, // hcc
            const std::complex<FPTYPE> *B, // scc
            const int m, // nbands
            FPTYPE *W, // eigenvalue
            std::complex<FPTYPE> *V)
    {
        using transpose_op = matrixTranspose_op<FPTYPE, psi::DEVICE_GPU>;
        using matrixset_op = matrixSetToAnother<FPTYPE, psi::DEVICE_GPU>;
        // init A_eigenvectors, transpose_B and all_W
        std::complex<FPTYPE> * A_eigenvectors = nullptr, * transpose_B = nullptr;
        if (nstart == ldh) {
            checkCudaErrors(hipMalloc((void **) &A_eigenvectors, sizeof(std::complex<FPTYPE>) * nstart * nstart));
            checkCudaErrors(hipMalloc((void **) &transpose_B, sizeof(std::complex<FPTYPE>) * nstart * nstart));

            transpose_op()(d, nstart, nstart, A,A_eigenvectors);
            transpose_op()(d, nstart, nstart, B, transpose_B);
        } else if (nstart < ldh) {
            // nstart < ldh
            checkCudaErrors(hipMalloc((void **) &A_eigenvectors, sizeof(std::complex<FPTYPE>) * nstart * nstart));
            checkCudaErrors(hipMalloc((void **) &transpose_B, sizeof(std::complex<FPTYPE>) * nstart * nstart));

            matrixset_op()(d, nstart, A, ldh,  A_eigenvectors, nstart);
            matrixset_op()(d, nstart, B, ldh, transpose_B, nstart);

            transpose_op()(d,nstart,nstart,A_eigenvectors,A_eigenvectors);
            transpose_op()(d,nstart,nstart,transpose_B,transpose_B);
        } else if (nstart > ldh) {
            assert(nstart < ldh);
        }

        FPTYPE * all_W = nullptr;
        checkCudaErrors(hipMalloc((void **) &all_W, sizeof(FPTYPE) * nstart));

        xhegvd_wrapper(HIPBLAS_FILL_MODE_LOWER, nstart, A_eigenvectors, nstart,
                       transpose_B, nstart, all_W);

        // get eigenvalues and eigenvectors.  only m !
        checkCudaErrors(hipMemcpy(W, all_W, sizeof(FPTYPE) * m, hipMemcpyDeviceToDevice));

        if (ldh == nstart) {
            transpose_op()(d, nstart, nstart, V, V);
            checkCudaErrors(
                    hipMemcpy(V, A_eigenvectors, sizeof(std::complex<FPTYPE>) * nstart * m, hipMemcpyDeviceToDevice));
            transpose_op()(d, nstart, nstart, V, V);
        } else {
            transpose_op()(d, ldh, ldh, V, V);
            matrixset_op()(d, m, A_eigenvectors, nstart, V, ldh);
            transpose_op()(d, ldh, ldh, V, V);
        }
        // free resources and destroy
        checkCudaErrors(hipFree(A_eigenvectors));
        checkCudaErrors(hipFree(transpose_B));
        checkCudaErrors(hipFree(all_W));
    }
};

template <>
void dngv_op<double, psi::DEVICE_GPU>::operator()(const psi::DEVICE_GPU* d,
                                                  const int nstart,
                                                  const int ldh,
                                                  const std::complex<double>* A,
                                                  const std::complex<double>* B,
                                                  double* W,
                                                  std::complex<double>* V)
{
    assert(nstart == ldh);
    // init A_eigenvectors & transpose_B
    double2 *A_eigenvectors, *transpose_B;
    checkCudaErrors(hipMalloc((void**)&A_eigenvectors, sizeof(double2) * ldh * nstart));
    checkCudaErrors(hipMalloc((void**)&transpose_B, sizeof(double2) * ldh * nstart));

    // transpose A, B  to A_eigenvectors, transpose_B
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, nstart, A, (std::complex<double>*)A_eigenvectors);
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, nstart, B, (std::complex<double>*)transpose_B);

    // init all_W
    double* all_W;
    checkCudaErrors(hipMalloc((void**)&all_W, sizeof(double) * ldh));

    // prepare some values for hipsolverDnZhegvd_bufferSize
    hipsolverHandle_t cusolverH;
    cusolverErrcheck(hipsolverDnCreate(&cusolverH));
    int* devInfo;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    int lwork = 0;
    cusolverErrcheck(hipsolverDnZhegvd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_UPPER,
        ldh,
        A_eigenvectors,
        nstart,
        transpose_B,
        nstart,
        all_W,
        &lwork));

    // allocate memery
    hipDoubleComplex* d_work;
    checkCudaErrors(hipMalloc((void**)&d_work, sizeof(hipDoubleComplex) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZhegvd(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_UPPER,
        ldh,
        A_eigenvectors,
        nstart,
        transpose_B,
        nstart,
        all_W,
        d_work,
        lwork,
        devInfo));

    checkCudaErrors(hipDeviceSynchronize());

    // get all eigenvalues and eigenvectors.
    checkCudaErrors(hipMemcpy(W, all_W, sizeof(double) * ldh, hipMemcpyDeviceToDevice));
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, ldh, nstart, (std::complex<double>*)A_eigenvectors, V);

    int info_gpu;
    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);

    // free the buffer
    checkCudaErrors(hipFree(d_work));
    // free resources and destroy
    checkCudaErrors(hipFree(A_eigenvectors));
    checkCudaErrors(hipFree(transpose_B));
    checkCudaErrors(hipFree(all_W));
    checkCudaErrors(hipFree(devInfo));
    cusolverErrcheck(hipsolverDnDestroy(cusolverH));
}

template <typename FPTYPE>
struct dngvd_op<FPTYPE, psi::DEVICE_GPU> {
    void operator()(
            const psi::DEVICE_GPU *d,
            const int nstart,
            const int ldh,
            const std::complex<FPTYPE> *A, // hcc
            const std::complex<FPTYPE> *B, // scc
            FPTYPE *W, // eigenvalue
            std::complex<FPTYPE> *V)
    {
        assert(nstart == ldh);
        // A to V
        checkCudaErrors(hipMemcpy(V, A, sizeof(std::complex<FPTYPE>) * ldh * nstart, hipMemcpyDeviceToDevice));

        xhegvd_wrapper(HIPBLAS_FILL_MODE_UPPER, nstart, V, ldh,
                       (std::complex<FPTYPE> *)B, ldh, W);
    }
};

template <typename FPTYPE>
struct dnevx_op<FPTYPE, psi::DEVICE_GPU> {
    void operator()(
            const psi::DEVICE_GPU *d,
            const int nstart,
            const int ldh,
            const std::complex<FPTYPE> *A, // hcc
            const int m,
            FPTYPE *W, // eigenvalue
            std::complex<FPTYPE> *V)
    {
        assert(nstart <= ldh);

        // A to V
        checkCudaErrors(hipMemcpy(V, A, sizeof(double2) * nstart * ldh, hipMemcpyDeviceToDevice));

        xheevd_wrapper(HIPBLAS_FILL_MODE_LOWER, nstart, V, nstart, W);

        // get eigenvalues and eigenvectors.  only m !
        checkCudaErrors(hipMemcpy(W, all_W, sizeof(FPTYPE) * m, hipMemcpyDeviceToDevice));

        if (ldh == nstart) {
            transpose_op()(d, nstart, nstart, V, V);
            checkCudaErrors(
                    hipMemcpy(V, A_eigenvectors, sizeof(std::complex<FPTYPE>) * nstart * m, hipMemcpyDeviceToDevice));
            transpose_op()(d, nstart, nstart, V, V);
        } else {
            transpose_op()(d, ldh, ldh, V, V);
            matrixset_op()(d, m, A_eigenvectors, nstart, V, ldh);
            transpose_op()(d, ldh, ldh, V, V);
        }

        // free resources and destroy
        checkCudaErrors(hipFree(A_eigenvectors));
        checkCudaErrors(hipFree(all_W));
    }
};

template struct dngvx_op<float, psi::DEVICE_GPU>;
template struct dngvd_op<float, psi::DEVICE_GPU>;
template struct dnevx_op<float, psi::DEVICE_GPU>;
template struct dngvx_op<double, psi::DEVICE_GPU>;
template struct dngvd_op<double, psi::DEVICE_GPU>;
template struct dnevx_op<double, psi::DEVICE_GPU>;

} // namespace hsolver