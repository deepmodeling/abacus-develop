#include "hip/hip_runtime.h"
#include "module_hsolver/include/dngvd_op.h"
#include "src_pdiag/hip/hip_runtime_api.h"

#include <hipsolver.h>

#define cusolverErrcheck(res)                      \
    {                                              \
        cusolverAssert((res), __FILE__, __LINE__); \
    }

// cuSOLVER API errors
static const char* _cusolverGetErrorEnum(hipsolverStatus_t error)
{
    switch (error)
    {
    case HIPSOLVER_STATUS_SUCCESS:
        return "HIPSOLVER_STATUS_SUCCESS";
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
        return "HIPSOLVER_STATUS_NOT_INITIALIZED";
    case HIPSOLVER_STATUS_ALLOC_FAILED:
        return "HIPSOLVER_STATUS_ALLOC_FAILED";
    case HIPSOLVER_STATUS_INVALID_VALUE:
        return "HIPSOLVER_STATUS_INVALID_VALUE";
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
        return "HIPSOLVER_STATUS_ARCH_MISMATCH";
    case HIPSOLVER_STATUS_MAPPING_ERROR:
        return "HIPSOLVER_STATUS_MAPPING_ERROR";
    case HIPSOLVER_STATUS_EXECUTION_FAILED:
        return "HIPSOLVER_STATUS_EXECUTION_FAILED";
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
        return "HIPSOLVER_STATUS_INTERNAL_ERROR";
    case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSOLVER_STATUS_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_NOT_SUPPORTED ";
    case HIPSOLVER_STATUS_ZERO_PIVOT:
        return "HIPSOLVER_STATUS_ZERO_PIVOT";
    case CUSOLVER_STATUS_INVALID_LICENSE:
        return "CUSOLVER_STATUS_INVALID_LICENSE";
    }
    return "<unknown>";
}

inline void cusolverAssert(hipsolverStatus_t code, const char* file, int line, bool abort = true)
{
    if (code != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuSOLVER Assert: %s %s %d\n", _cusolverGetErrorEnum(code), file, line);
        if (abort)
            exit(code);
    }
}

namespace hsolver
{


static hipsolverHandle_t cusolverH = nullptr;


void createCUSOLVERhandle(){
    if (cusolverH == nullptr) {
        cusolverErrcheck(hipsolverDnCreate(&cusolverH));
    }
}

void destoryCUSOLVERhandle(){
    if (cusolverH != nullptr) {
        cusolverErrcheck(hipsolverDnDestroy(cusolverH));
        cusolverH = nullptr;
    }
}

template <>
void dngvx_op<double, psi::DEVICE_GPU>::operator()(const psi::DEVICE_GPU* d,
                                                   const int row,
                                                   const int col,
                                                   const std::complex<double>* A,
                                                   const std::complex<double>* B,
                                                   const int m,
                                                   double* W,
                                                   std::complex<double>* V)
{
    createBLAShandle();

    // init A_eigenvectors, transpose_B and all_W
    double2 *A_eigenvectors, *transpose_B;
    if (row == col)
    {
        checkCudaErrors(hipMalloc((void**)&A_eigenvectors, sizeof(double2) * row * col));
        checkCudaErrors(hipMalloc((void**)&transpose_B, sizeof(double2) * row * col));

        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, col, row, A, (std::complex<double>*)A_eigenvectors);
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, col, row, B, (std::complex<double>*)transpose_B);
    } else
    {
        // row max
        // col min
        checkCudaErrors(hipMalloc((void**)&A_eigenvectors, sizeof(double2) * col * col));
        checkCudaErrors(hipMalloc((void**)&transpose_B, sizeof(double2) * col * col));

        matrixSetToAnother<double, psi::DEVICE_GPU>()(d, col, A, row, (std::complex<double>*)A_eigenvectors, col);
        matrixSetToAnother<double, psi::DEVICE_GPU>()(d, col, B, row, (std::complex<double>*)transpose_B, col);

        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, col, col, (std::complex<double>*)A_eigenvectors, (std::complex<double>*)A_eigenvectors);
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, col, col, (std::complex<double>*)transpose_B, (std::complex<double>*)transpose_B);
    }

    double* all_W;
    checkCudaErrors(hipMalloc((void**)&all_W, sizeof(double) * col));

    // prepare some values for hipsolverDnZhegvd_bufferSize
    hipsolverHandle_t cusolverH;
    cusolverErrcheck(hipsolverDnCreate(&cusolverH));
    int* devInfo;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    int lwork = 0;
    cusolverErrcheck(hipsolverDnZhegvd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_LOWER,
        col,
        A_eigenvectors,
        col,
        transpose_B,
        col,
        all_W,
        &lwork));

    // allocate memery
    hipDoubleComplex* d_work;
    checkCudaErrors(hipMalloc((void**)&d_work, sizeof(hipDoubleComplex) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZhegvd(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_LOWER,
        col,
        A_eigenvectors,
        col,
        transpose_B,
        col,
        all_W,
        d_work,
        lwork,
        devInfo));

    checkCudaErrors(hipDeviceSynchronize());

    // get eigenvalues and eigenvectors.  only m !
    checkCudaErrors(hipMemcpy(W, all_W, sizeof(double) * m, hipMemcpyDeviceToDevice));

    if (row == col)
    {
        checkCudaErrors(hipMemcpy(V, A_eigenvectors, sizeof(std::complex<double>) * col * m, hipMemcpyDeviceToDevice));

        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, col, row, V, V);
    } else
    {
        matrixTranspose_op<double, psi::DEVICE_GPU>()(d, col, col, (std::complex<double>*)A_eigenvectors, (std::complex<double>*)A_eigenvectors);

        matrixSetToAnother<double, psi::DEVICE_GPU>()(d, col, (std::complex<double>*)A_eigenvectors, col, V, row);
    }
    

    int info_gpu;
    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);

    // free the buffer
    checkCudaErrors(hipFree(d_work));
    // free resources and destroy
    checkCudaErrors(hipFree(A_eigenvectors));
    checkCudaErrors(hipFree(all_W));
    checkCudaErrors(hipFree(devInfo));
    cusolverErrcheck(hipsolverDnDestroy(cusolverH));

    destoryBLAShandle();
}

template <>
void dngv_op<double, psi::DEVICE_GPU>::operator()(const psi::DEVICE_GPU* d,
                                                  const int row,
                                                  const int col,
                                                  const std::complex<double>* A,
                                                  const std::complex<double>* B,
                                                  double* W,
                                                  std::complex<double>* V)
{
    createBLAShandle();

    // init A_eigenvectors & transpose_B
    double2 *A_eigenvectors, *transpose_B;
    checkCudaErrors(hipMalloc((void**)&A_eigenvectors, sizeof(double2) * row * col));
    checkCudaErrors(hipMalloc((void**)&transpose_B, sizeof(double2) * row * col));

    // transpose A, B  to A_eigenvectors, transpose_B
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, A, (std::complex<double>*)A_eigenvectors);
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, B, (std::complex<double>*)transpose_B);

    // init all_W
    double* all_W;
    checkCudaErrors(hipMalloc((void**)&all_W, sizeof(double) * row));

    // prepare some values for hipsolverDnZhegvd_bufferSize
    hipsolverHandle_t cusolverH;
    cusolverErrcheck(hipsolverDnCreate(&cusolverH));
    int* devInfo;
    checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

    // calculate the sizes needed for pre-allocated buffer.
    int lwork = 0;
    cusolverErrcheck(hipsolverDnZhegvd_bufferSize(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_UPPER,
        row,
        A_eigenvectors,
        col,
        transpose_B,
        col,
        all_W,
        &lwork));

    // allocate memery
    hipDoubleComplex* d_work;
    checkCudaErrors(hipMalloc((void**)&d_work, sizeof(hipDoubleComplex) * lwork));

    // compute eigenvalues and eigenvectors.
    cusolverErrcheck(hipsolverDnZhegvd(
        cusolverH,
        HIPSOLVER_EIG_TYPE_1, // itype = HIPSOLVER_EIG_TYPE_1: A*x = (lambda)*B*x.
        HIPSOLVER_EIG_MODE_VECTOR, // jobz = HIPSOLVER_EIG_MODE_VECTOR : Compute eigenvalues and eigenvectors.
        HIPBLAS_FILL_MODE_UPPER,
        row,
        A_eigenvectors,
        col,
        transpose_B,
        col,
        all_W,
        d_work,
        lwork,
        devInfo));

    checkCudaErrors(hipDeviceSynchronize());

    // get all eigenvalues and eigenvectors.
    checkCudaErrors(hipMemcpy(W, all_W, sizeof(double) * row, hipMemcpyDeviceToDevice));
    matrixTranspose_op<double, psi::DEVICE_GPU>()(d, row, col, (std::complex<double>*)A_eigenvectors, V);

    int info_gpu;
    checkCudaErrors(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    assert(0 == info_gpu);

    // free the buffer
    checkCudaErrors(hipFree(d_work));
    // free resources and destroy
    checkCudaErrors(hipFree(A_eigenvectors));
    checkCudaErrors(hipFree(all_W));
    checkCudaErrors(hipFree(devInfo));
    cusolverErrcheck(hipsolverDnDestroy(cusolverH));

    destoryBLAShandle();
}

} // namespace hsolver