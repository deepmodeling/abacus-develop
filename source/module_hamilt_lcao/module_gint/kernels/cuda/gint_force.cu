#include "hip/hip_runtime.h"
#include "gint_force.cuh"
#include "interp.cuh"
#include "module_hamilt_lcao/module_gint/gint_force_gpu.h"
#include "module_hamilt_lcao/module_gint/kernels/cuda/cuda_tools.cuh"
#include "module_hamilt_lcao/module_gint/kernels/cuda/gint_force.cuh"
#include "module_hamilt_lcao/module_gint/kernels/cuda/sph.cuh"
#include "hip/hip_runtime.h"
// CUDA kernel to calculate psi and force
namespace GintKernel
{

/*!
 * \file
 * \brief CUDA kernel to calculate psi and force
 *
 * CUDA kernel that performs calculations on psi and force.
 *
 * \param ylmcoef Pointer to the Ylm coefficients
 * \param delta_r_g Delta r value
 * \param bxyz_g Bxyz value
 * \param nwmax_g Nwmax value
 * \param input_double Array of double input values
 * \param input_int Array of int input values
 * \param num_psir Array containing the number of psi for each block
 * \param psi_size_max Maximum size of psi
 * \param ucell_atom_nwl Array containing Ucell atom nwl values
 * \param atom_iw2_new Array indicating whether atom_iw2 is new
 * \param atom_iw2_ylm Array of atom_iw2 Ylm values
 * \param atom_iw2_l Array of atom_iw2 l values
 * \param atom_nw Array of atom_nw values
 * \param nr_max Maximum nr value
 * \param psi_u Array for psi_u values,recording the Spherical Harmonics from psi
 * \param psir_r Array for psir_r values,recored the distance from psi
 * \param psir_lx Array for psir_lx values,recored the force left in x
 * \param psir_ly Array for psir_ly values,recored the force left in y
 * \param psir_lz Array for psir_lz values,recored the force left in z
 * \param psir_lxx Array for psir_lxx values,recored the stress left in xx
 * \param psir_lxy Array for psir_lxy values,recored the stress left in xy
 * \param psir_lxz Array for psir_lxz values,recored the stress left in xz
 * \param psir_lyy Array for psir_lyy values,recored the stress left in yy
 * \param psir_lyz Array for psir_lyz values,recored the stress left in yz
 * \param psir_lzz Array for psir_lzz values,recored the stress left in zz
 */

__global__ void get_psi_force(double* ylmcoef,
                              double delta_r_g,
                              int bxyz_g,
                              double nwmax_g,
                              double* input_double,
                              int* input_int,
                              int* num_psir,
                              int psi_size_max,
                              int* ucell_atom_nwl,
                              bool* atom_iw2_new,
                              int* atom_iw2_ylm,
                              int* atom_iw2_l,
                              int* atom_nw,
                              int nr_max,
                              double* psi_u,
                              double* psir_r,
                              double* psir_lx,
                              double* psir_ly,
                              double* psir_lz,
                              double* psir_lxx,
                              double* psir_lxy,
                              double* psir_lxz,
                              double* psir_lyy,
                              double* psir_lyz,
                              double* psir_lzz)
{
    // Get the size of psi for the current block
    int size = num_psir[blockIdx.x];
    int start_index = psi_size_max * blockIdx.x;
    int end_index = start_index + size;
    start_index += threadIdx.x + blockDim.x * blockIdx.y;
    // Loop over the psi indices for the current block
    for (int index = start_index; index < end_index;
         index += blockDim.x * gridDim.y)
    {
        // Extract information from input arrays
        double dr[3];
        int index_double = index * 5;
        dr[0] = input_double[index_double];
        dr[1] = input_double[index_double + 1];
        dr[2] = input_double[index_double + 2];
        double distance = input_double[index_double + 3];
        distance = distance * distance;
        double vlbr3_value = input_double[index_double + 4];
        // begin calculation
        double ylma[49]; // Attention!!! At present, we only use L=5 at
                         // most. So (L+1) * (L+1)=36
        double grly[49][3];
        int index_int = index * 2;
        int it = input_int[index_int];
        int dist_tmp = input_int[index_int + 1];

        int nwl = ucell_atom_nwl[it];
        spherical_harmonics_d(dr, distance, grly, nwl, ylma, ylmcoef);

        interpolate_f(distance,
                      delta_r_g,
                      it,
                      nwmax_g,
                      nr_max,
                      atom_nw,
                      atom_iw2_new,
                      psi_u,
                      atom_iw2_l,
                      atom_iw2_ylm,
                      psir_r,
                      dist_tmp,
                      ylma,
                      vlbr3_value,
                      psir_lx,
                      dr,
                      grly,
                      psir_ly,
                      psir_lz,
                      psir_lxx,
                      psir_lxy,
                      psir_lxz,
                      psir_lyy,
                      psir_lyz,
                      psir_lzz);
    }
}


/**
 * \brief Compute dot product of stress components and partial derivatives.
 *
 * This CUDA kernel computes the dot product of stress components and partial
 * derivatives based on the input arrays.
 *
 * \param psir_lxx Array of psir_lxx values.
 * \param psir_lxy Array of psir_lxy values.
 * \param psir_lxz Array of psir_lxz values.
 * \param psir_lyy Array of psir_lyy values.
 * \param psir_lyz Array of psir_lyz values.
 * \param psir_lzz Array of psir_lzz values.
 * \param psir_ylm_dm Array of psir_ylm_dm values.
 * \param stress_dot Output array for the dot product of stress components.
 * \param elements_num Number of elements in the input arrays.
 */

__global__ void dot_product_stress(double* psir_lxx,
                                   double* psir_lxy,
                                   double* psir_lxz,
                                   double* psir_lyy,
                                   double* psir_lyz,
                                   double* psir_lzz,
                                   double* psir_ylm_dm,
                                   double* stress_dot,
                                   int elements_num)
{

    __shared__ double cache[256][6]; 
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    double tmp[6] = {0.0};
    while (tid < elements_num)
    {
        tmp[0] += psir_lxx[tid] * psir_ylm_dm[tid] * 2;
        tmp[1] += psir_lxy[tid] * psir_ylm_dm[tid] * 2;
        tmp[2] += psir_lxz[tid] * psir_ylm_dm[tid] * 2;
        tmp[3] += psir_lyy[tid] * psir_ylm_dm[tid] * 2;
        tmp[4] += psir_lyz[tid] * psir_ylm_dm[tid] * 2;
        tmp[5] += psir_lzz[tid] * psir_ylm_dm[tid] * 2;
        tid += blockDim.x * gridDim.x;
    }

    for (int i = 0; i < 6; i++)
    {
        cache[cacheIndex][i] = tmp[i];
    }
    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0)
    {
        if (cacheIndex < i)
        {
            for (int index = 0; index < 6; index++)
            {
                cache[cacheIndex][index] += cache[cacheIndex + i][index];
            }
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0){
        for (int index = 0; index < 6; index++)
        {
            atomicAdd(&stress_dot[index], cache[0][index]); // Use atomicAdd() instead of atomic_add().
            // stress_dot[blockIdx.x + gridDim.x * index] = cache[0][index];
        }
    }
}

/**
 * @brief Calculate the dot product force.
 *
 * This function calculates the dot product force based on the provided
 * parameters.
 *
 * @param psir_lx Pointer to the array of psir_lx values.
 * @param psir_ly Pointer to the array of psir_ly values.
 * @param psir_lz Pointer to the array of psir_lz values.
 * @param psir_ylm_dm Pointer to the array of psir_ylm_dm values.
 * @param force_dot Pointer to the array where the calculated force will be
 * stored.
 * @param iat Pointer to the array of iat values.
 * @param nwmax Maximum value for nwmax.
 * @param max_size Maximum size for arrays.
 * @param elements_num Number of elements to process.
 */

__global__ void dot_product_force(double* psir_lx,
                                  double* psir_ly,
                                  double* psir_lz,
                                  double* psir_ylm_dm,
                                  double* force_dot,
                                  int* iat,
                                  int nwmax,
                                  int max_size,
                                  int elements_num)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < elements_num)
    {
        int iat_on_nbz = iat[tid];
        if (iat_on_nbz <= -1)
        {
            tid += blockDim.x * gridDim.x;
            continue;
        }

        int iat_index = tid * 3;
        int dist = tid * nwmax;
        double tmp[3] = {0.0};

        for (int i = 0; i < nwmax; i++)
        {
            tmp[0] += psir_lx[dist + i] * psir_ylm_dm[dist + i] * 2;
            tmp[1] += psir_ly[dist + i] * psir_ylm_dm[dist + i] * 2;
            tmp[2] += psir_lz[dist + i] * psir_ylm_dm[dist + i] * 2;
        }
        
        for (int i = 0; i < 3; i++)
        {
            atomicAdd(&force_dot[iat_on_nbz*3 + i], tmp[i]);
        }
        tid += blockDim.x * gridDim.x;
    }
}
} // namespace GintKernel
