#include "hip/hip_runtime.h"
#include "omp.h"
#include "gint_tools.h"
#include "gint_vl.h"
#include "module_base/ylm.h"
#include "module_basis/module_ao/ORB_read.h"
#include "module_hamilt_pw/hamilt_pwdft/global.h"
#include <fstream>
#include <sstream>

__constant__ double ylmcoef[36];
__constant__ int bx_g[1];
__constant__ int by_g[1];
__constant__ int bz_g[1];
__constant__ int bxyz_g[1];
__constant__ int max_size_g[1];
__constant__ int nwmax_g[1];
__constant__ int namax_g[1];
__constant__ int nnnmax_g[1];
__constant__ int ntype_g[1];
__constant__ double delta_r_g[1];
__constant__ double vfactor_g[1];

void dump_cuda_array_to_file(double * cuda_array, int width, int hight, const std::string & filename)
{
    double *h_data = new double[width * hight];
    hipMemcpy(h_data, cuda_array, width * hight * sizeof(double), hipMemcpyDeviceToHost);

    std::ofstream outFile(filename);
    if (!outFile.is_open()) {
        std::cerr << "Failed to open file for writing." << std::endl;
    }
    for (int j = 0; j < hight; ++j) {
        for (int i = 0; i < width; ++i) {
            outFile << "hight" << j << "   width:" << i << "   "<< h_data[j * width + i] << std::endl;
        }
    }
    outFile.close();
    delete[] h_data;
}

__global__ void get_psi(double *dr_all,
                        int *it_all,
                        int *psir_ylm_start,
                        int *num_psir,
                        int psi_size_up,
                        bool *cal_flag,
                        int *ucell_atom_nwl,
                        bool *atom_iw2_new,
                        int *atom_iw2_ylm,
                        int *atom_nw,
                        int nr_max,
                        double *psi_u,
                        double *dpsi_u,
                        double *psir_ylm)
{
    // int grid_index_now = blockIdx.x;
    int size = num_psir[blockIdx.x];
    int loop_size = size / blockDim.x;

    if (loop_size != 0)
    {
        int start_index = loop_size * threadIdx.x + psi_size_up * blockIdx.x;
        int end_index = start_index + loop_size;
        for (int index = start_index; index < end_index; index++)
        {
            int it = it_all[index];
            if (it < 0) continue;

            double dr[3];
            dr[0] = dr_all[index * 4];
            dr[1] = dr_all[index * 4 + 1];
            dr[2] = dr_all[index * 4 + 2];
            double distance = dr_all[index * 4 + 3];
            int dist_tmp = psir_ylm_start[index];

            cal_flag[dist_tmp] = true;

            // begin calculation
            double ylma[150];
            /***************************
            L = 0
            ***************************/
            ylma[0] = ylmcoef[0]; // l=0, m=0
            int nwl = ucell_atom_nwl[it];

            if (nwl == 0)
                goto YLM_END;

            /***************************
            L = 1
            ***************************/
            ylma[1] = ylmcoef[1] * dr[2];  // l=1, m=0
            ylma[2] = -ylmcoef[1] * dr[0]; // l=1, m=1
            ylma[3] = -ylmcoef[1] * dr[1]; // l=1, m=-1
            if (nwl == 1)
                goto YLM_END;

            /***************************
            L = 2
            ***************************/
            ylma[4] = ylmcoef[2] * dr[2] * ylma[1] - ylmcoef[3] * ylma[0]; // l=2, m=0
            {
                double tmp0 = ylmcoef[4] * dr[2];
                ylma[5] = tmp0 * ylma[2]; // l=2,m=1
                ylma[6] = tmp0 * ylma[3]; // l=2,m=-1

                tmp0 = ylmcoef[4] * dr[0];
                ylma[7] = ylmcoef[5] * ylma[4] - ylmcoef[6] * ylma[0] - tmp0 * ylma[2]; // l=2,m=2
                ylma[8] = -tmp0 * ylma[3];
                //	ylma[grid_index*nnnmax+8] = tmp1+tmp2*ylma[grid_index*nnnmax+3];//l=2,m=-2
                if (nwl == 2)
                    goto YLM_END;

                /***************************
                L = 3
                ***************************/
                ylma[9] = ylmcoef[7] * dr[2] * ylma[4] - ylmcoef[8] * ylma[1]; // l=3, m=0

                tmp0 = ylmcoef[9] * dr[2];
                ylma[10] = tmp0 * ylma[5] - ylmcoef[10] * ylma[2]; // l=3,m=1
                ylma[11] = tmp0 * ylma[6] - ylmcoef[10] * ylma[3]; // l=3,m=-1

                tmp0 = ylmcoef[11] * dr[2];
                ylma[12] = tmp0 * ylma[7]; // l=3,m=2
                ylma[13] = tmp0 * ylma[8]; // l=3,m=-2

                tmp0 = ylmcoef[14] * dr[0];
                ylma[14] = ylmcoef[12] * ylma[10] - ylmcoef[13] * ylma[2] - tmp0 * ylma[7]; // l=3,m=3
                ylma[15] = ylmcoef[12] * ylma[11] - ylmcoef[13] * ylma[3] - tmp0 * ylma[8]; // l=3,m=-3
                if (nwl == 3)
                    goto YLM_END;

                /***************************
                L = 4
                ***************************/
                ylma[16] = ylmcoef[15] * dr[2] * ylma[9] - ylmcoef[16] * ylma[4]; // l=4,m=0

                tmp0 = ylmcoef[17] * dr[2];
                ylma[17] = tmp0 * ylma[10] - ylmcoef[18] * ylma[5]; // l=4,m=1
                ylma[18] = tmp0 * ylma[11] - ylmcoef[18] * ylma[6]; // l=4,m=-1

                tmp0 = ylmcoef[19] * dr[2];
                ylma[19] = tmp0 * ylma[12] - ylmcoef[20] * ylma[7]; // l=4,m=2
                ylma[20] = tmp0 * ylma[13] - ylmcoef[20] * ylma[8]; // l=4,m=-2

                tmp0 = 3.0 * dr[2];
                ylma[21] = tmp0 * ylma[14]; // l=4,m=3
                ylma[22] = tmp0 * ylma[15]; // l=4,m=-3

                tmp0 = ylmcoef[23] * dr[0];
                ylma[23] = ylmcoef[21] * ylma[19] - ylmcoef[22] * ylma[7] - tmp0 * ylma[14]; // l=4,m=4
                ylma[24] = ylmcoef[21] * ylma[20] - ylmcoef[22] * ylma[8] - tmp0 * ylma[15]; // l=4,m=-4
                if (nwl == 4)
                    goto YLM_END;

                /***************************
                L = 5
                ***************************/
                ylma[25] = ylmcoef[24] * dr[2] * ylma[16] - ylmcoef[25] * ylma[9]; // l=5,m=0

                tmp0 = ylmcoef[26] * dr[2];
                ylma[26] = tmp0 * ylma[17] - ylmcoef[27] * ylma[10]; // l=5,m=1
                ylma[27] = tmp0 * ylma[18] - ylmcoef[27] * ylma[11]; // l=5,m=-1

                tmp0 = ylmcoef[28] * dr[2];
                ylma[28] = tmp0 * ylma[19] - ylmcoef[29] * ylma[12]; // l=5,m=2
                ylma[29] = tmp0 * ylma[20] - ylmcoef[29] * ylma[13]; // l=5,m=-2

                tmp0 = ylmcoef[30] * dr[2];
                ylma[30] = tmp0 * ylma[21] - ylmcoef[31] * ylma[14]; // l=5,m=3
                ylma[31] = tmp0 * ylma[22] - ylmcoef[31] * ylma[15]; // l=5,m=-3

                tmp0 = ylmcoef[32] * dr[2];
                ylma[32] = tmp0 * ylma[23]; // l=5,m=4
                ylma[33] = tmp0 * ylma[24]; // l=5,m=-4

                tmp0 = ylmcoef[35] * dr[0];
                ylma[34] = ylmcoef[33] * ylma[30] - ylmcoef[34] * ylma[14] - tmp0 * ylma[23]; // l=5,m=5
                ylma[35] = ylmcoef[33] * ylma[31] - ylmcoef[34] * ylma[15] - tmp0 * ylma[24]; // l=5,m=-5
                if (nwl == 5)
                    goto YLM_END;

                // if nwl > 5
                for (int il = 6; il <= nwl; il++)
                {
                    int istart = il * il;
                    int istart1 = (il - 1) * (il - 1);
                    int istart2 = (il - 2) * (il - 2);

                    double fac2 = sqrt(4.0 * istart - 1.0);
                    double fac4 = sqrt(4.0 * istart1 - 1.0);

                    for (int im = 0; im < 2 * il - 1; im++)
                    {
                        int imm = (im + 1) / 2;
                        //			if (im % 2 == 0) imm *= -1;

                        ylma[istart + im] = fac2 / sqrt((double)istart - imm * imm) * (dr[2] * ylma[istart1 + im] - sqrt((double)istart1 - imm * imm) / fac4 * ylma[istart2 + im]);
                    }

                    double bl1 = sqrt(2.0 * il / (2.0 * il + 1.0));
                    double bl2 = sqrt((2.0 * il - 2.0) / (2.0 * il - 1.0));
                    double bl3 = sqrt(2.0) / fac2;

                    ylma[istart + 2 * il - 1] = (bl3 * ylma[istart + 2 * il - 5] - bl2 * ylma[istart2 + 2 * il - 5] - 2.0 * dr[0] * ylma[istart1 + 2 * il - 3]) / bl1;
                    ylma[istart + 2 * il] = (bl3 * ylma[istart + 2 * il - 4] - bl2 * ylma[istart2 + 2 * il - 4] - 2.0 * dr[0] * ylma[istart1 + 2 * il - 2]) / bl1;
                }
            }
        YLM_END:
            distance /= delta_r_g[0];

            int ip = (int)(distance);
            double dx = distance - ip;
            double dx2 = dx * dx;
            double dx3 = dx2 * dx;

            double c3 = 3.0 * dx2 - 2.0 * dx3;
            double c1 = 1.0 - c3;
            double c2 = (dx - 2.0 * dx2 + dx3) * delta_r_g[0];
            double c4 = (dx3 - dx2) * delta_r_g[0];

            int iw;
            double phi = 0.0;
            int it_nw = it * nwmax_g[0];
            const int it_nw_nr_ip = it_nw * nr_max + ip;
            int iw_nr = it_nw_nr_ip;
            dist_tmp = dist_tmp * nwmax_g[0];
            for (iw = 0; iw < atom_nw[it]; ++iw)
            {
                int it_nw_iw = it_nw + iw;
                if (atom_iw2_new[it_nw_iw])
                {
                    phi = c1 * psi_u[iw_nr] + c2 * dpsi_u[iw_nr] + c3 * psi_u[iw_nr + 1] + c4 * dpsi_u[iw_nr + 1];
                }
                psir_ylm[dist_tmp + iw] = phi * ylma[atom_iw2_ylm[it_nw_iw]];
                iw_nr += nr_max;
            }
        }
    } // if size
}
__global__ void get_vldr3(int ij_index,
                          int nbx,
                          int nby,
                          int nbz,
                          int nbz_start,
                          int ncy,
                          int nczp,
                          double *vlocal,
                          int *start_ind_g,
                          double *vldr3)
{
    int k = blockIdx.x;
    int ii = threadIdx.x;
    int jj = threadIdx.y;
    int kk = threadIdx.z;
    int vindex = ii * ncy * nczp + jj * nczp + kk + start_ind_g[ij_index + k];
    vldr3[k * bx_g[0] * by_g[0] * bz_g[0] + ii * by_g[0] * bz_g[0] + jj * bz_g[0] + kk] = vlocal[vindex] * vfactor_g[0];
}

__global__ void psi_multiple(int grid_index,
                             int *how_many_atoms,
                             double *atom_pair_index1_g,
                             double *atom_pair_index2_g,
                             int *bcell_start,
                             int *which_atom,
                             int *iat2it,
                             int *iat2ia,
                             int *itiaiw2iwt,
                             bool *cal_flag,
                             double *psir_ylm,
                             int *trace_lo,
                             int *atom_nw,
                             double *vldr3,
                             double *GridVlocal,
                             int lgd)
{
    int atomnow1 = blockIdx.x;
    int atomnow2 = blockIdx.y;
    int k = blockIdx.z;
    grid_index += k;
    int iw1 = threadIdx.x;
    int iw2 = threadIdx.y;
    if (atomnow1 >= how_many_atoms[grid_index] || atomnow2 >= how_many_atoms[grid_index])
    {
        return;
    }
    int iat1 = which_atom[bcell_start[grid_index] + atomnow1];
    int iat2 = which_atom[bcell_start[grid_index] + atomnow2];
    int it1 = iat2it[iat1];
    int it2 = iat2it[iat2];
    if (iw1 >= atom_nw[it1] || iw2 >= atom_nw[it2])
    {
        return;
    }

    int lo1 = trace_lo[itiaiw2iwt[it1 * namax_g[0] + iat2ia[iat1]]];
    int lo2 = trace_lo[itiaiw2iwt[it2 * namax_g[0] + iat2ia[iat2]]];
    if (lo1 <= lo2)
    {
        int lo1_iw1 = lo1 + iw1;
        int lo2_iw2 = lo2 + iw2;
        double v2 = 0.0;
        for (int ib = 0; ib < bxyz_g[0]; ++ib)
        {
            int vldr3_index = k * bxyz_g[0] + ib;
            int calc_index1 = vldr3_index * max_size_g[0];
            int calc_index2 = calc_index1 + atomnow2;
            calc_index1 += atomnow1;
            if (cal_flag[calc_index1] &&
                cal_flag[calc_index2])
            {
                v2 += psir_ylm[calc_index1 * nwmax_g[0] + iw1] * vldr3[vldr3_index] * psir_ylm[calc_index2 * nwmax_g[0] + iw2];
            }
        }
        atomicAdd(&(GridVlocal[lo1_iw1 * lgd + lo2_iw2]), v2);
    }
}

void gint_gamma_vl_gpu(double *GridVlocal_now,
                       const int lgd,
                       const int nnnmax,
                       const int max_size,
                       const double vfactor,
                       const double *vlocal,
                       const double *ylmcoef_now,
                       const int bx,
                       const int by,
                       const int bz,
                       const int bxyz,
                       const int ncx,
                       const int ncy,
                       const int nczp,
                       const int NLOCAL_now,
                       const int nbxx,
                       int *start_ind,
                       const Grid_Technique &GridT)
{
    // printf("\n**************START GPU SEG***************\n");

    hipEvent_t t1, t2, t3, t4;
    hipEventCreate(&t1);
    hipEventCreate(&t2);
    hipEventCreate(&t3);
    hipEventCreate(&t4);

    hipEventRecord(t1);

    const Numerical_Orbital_Lm *pointer;
    //const double delta_r = GlobalC::ORB.dr_uniform;
    //const int total_atoms_on_grid = GridT.total_atoms_on_grid;
    const int nbx = GridT.nbx;
    const int nby = GridT.nby;
    const int nbz_start = GridT.nbzp_start;
    const int nbz = GridT.nbzp;
    const int nwmax = GlobalC::ucell.nwmax;
    const int namax = GlobalC::ucell.namax;
    const int ntype = GlobalC::ucell.ntype;

    double max_cut = 0;
    for (int i = 0; i < ntype; i++)
    {
        if (GlobalC::ORB.Phi[i].getRcut() > max_cut)
        {
            max_cut = GlobalC::ORB.Phi[i].getRcut();
        }
    }

    int *atom_nw_now;
    int *ucell_atom_nwl_now;
    atom_nw_now = new int[ntype];
    ucell_atom_nwl_now = new int[ntype];
    for (int i = 0; i < ntype; i++)
    {
        atom_nw_now[i] = GlobalC::ucell.atoms[i].nw;
        ucell_atom_nwl_now[i] = GlobalC::ucell.atoms[i].nwl;
    }

    int nr_max = static_cast<int>(1000 * max_cut) + 10;
    double *psi_u_now = new double[ntype * nwmax * nr_max];
    double *dpsi_u_now = new double[ntype * nwmax * nr_max];
    bool *atom_iw2_new_now = new bool[ntype * nwmax];
    int *atom_iw2_ylm_now = new int[ntype * nwmax];

    Atom *atomx;
    for (int i = 0; i < ntype; i++)
    {
        atomx = &GlobalC::ucell.atoms[i];
        for (int j = 0; j < nwmax; j++)
        {
            if (j < atomx->nw)
            {

                atom_iw2_new_now[i * nwmax + j] = atomx->iw2_new[j];
                atom_iw2_ylm_now[i * nwmax + j] = atomx->iw2_ylm[j];
                pointer = &GlobalC::ORB.Phi[i].PhiLN(atomx->iw2l[j], atomx->iw2n[j]);
                for (int k = 0; k < nr_max; k++)
                {
                    if (k < pointer->nr_uniform)
                    {
                        psi_u_now[i * nwmax * nr_max + j * nr_max + k] = pointer->psi_uniform[k];
                        dpsi_u_now[i * nwmax * nr_max + j * nr_max + k] = pointer->dpsi_uniform[k];
                    }
                    else
                    {
                        psi_u_now[i * nwmax * nr_max + j * nr_max + k] = 0;
                        dpsi_u_now[i * nwmax * nr_max + j * nr_max + k] = 0;
                    }
                }
            }
            else
            {

                atom_iw2_new_now[i * nwmax + j] = false;
                atom_iw2_ylm_now[i * nwmax + j] = 0;
                for (int k = 0; k < nr_max; k++)
                {
                    psi_u_now[i * nwmax * nr_max + j * nr_max + k] = 0;
                    dpsi_u_now[i * nwmax * nr_max + j * nr_max + k] = 0;
                }
            }
        }
    }

    size_t size_itiaiw2iwt = ntype * namax;
    int *itiaiw2iwt_now = new int[size_itiaiw2iwt];
    for (int i = 0; i < ntype; i++)
    {
        for (int j = 0; j < namax; j++)
        {
            itiaiw2iwt_now[i * namax + j] = GlobalC::ucell.itiaiw2iwt(i, j, 0);
        }
    }

    hipMemcpyToSymbol(HIP_SYMBOL(ylmcoef), ylmcoef_now, 36 * sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(bx_g), &bx, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(by_g), &by, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(bz_g), &bz, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(bxyz_g), &bxyz, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(max_size_g), &max_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(nwmax_g), &nwmax, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(namax_g), &namax, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(nnnmax_g), &nnnmax, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(ntype_g), &ntype, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(delta_r_g), &GlobalC::ORB.dr_uniform, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(vfactor_g), &vfactor, sizeof(double));

    // read only
    int *how_many_atoms;
    hipError_t status = hipMalloc((void **)&how_many_atoms, nbx * nby * nbz * sizeof(int));
    if (status != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(status));
        return;
    }

    hipError_t status2 = hipMemcpy(how_many_atoms, GridT.how_many_atoms, nbx * nby * nbz * sizeof(int), hipMemcpyHostToDevice);
    if (status2 != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(status2));
        return;
    }

    int *bcell_start;
    hipMalloc((void **)&bcell_start, nbx * nby * nbz * sizeof(int));
    hipMemcpy(bcell_start, GridT.bcell_start, nbx * nby * nbz * sizeof(int), hipMemcpyHostToDevice);

    int *which_bigcell;
    hipMalloc((void **)&which_bigcell, GridT.total_atoms_on_grid * sizeof(int));
    hipMemcpy(which_bigcell, GridT.which_bigcell, GridT.total_atoms_on_grid * sizeof(int), hipMemcpyHostToDevice);

    int *which_atom;
    hipMalloc((void **)&which_atom, GridT.total_atoms_on_grid * sizeof(int));
    hipMemcpy(which_atom, GridT.which_atom, GridT.total_atoms_on_grid * sizeof(int), hipMemcpyHostToDevice);

    int *iat2it;
    size_t size_iat2it = GlobalC::ucell.nat;
    hipMalloc((void **)&iat2it, size_iat2it * sizeof(int));
    hipMemcpy(iat2it, GlobalC::ucell.iat2it, size_iat2it * sizeof(int), hipMemcpyHostToDevice);

    int *iat2ia;
    size_t size_iat2ia = GlobalC::ucell.nat;
    hipMalloc((void **)&iat2ia, size_iat2ia * sizeof(int));
    hipMemcpy(iat2ia, GlobalC::ucell.iat2ia, size_iat2ia * sizeof(int), hipMemcpyHostToDevice);

    double *vlocal_cu;
    hipMalloc((void **)&vlocal_cu, ncx * ncy * nczp * sizeof(double));
    hipMemcpy(vlocal_cu, vlocal, ncx * ncy * nczp * sizeof(double), hipMemcpyHostToDevice);

    int *atom_nw;
    hipMalloc((void **)&atom_nw, ntype * sizeof(int));
    hipMemcpy(atom_nw, atom_nw_now, ntype * sizeof(int), hipMemcpyHostToDevice);

    int *ucell_atom_nwl;
    hipMalloc((void **)&ucell_atom_nwl, ntype * sizeof(int));
    hipMemcpy(ucell_atom_nwl, ucell_atom_nwl_now, ntype * sizeof(int), hipMemcpyHostToDevice);

    double *psi_u;
    double *dpsi_u;
    hipMalloc((void **)&psi_u, ntype * nwmax * nr_max * sizeof(double));
    hipMalloc((void **)&dpsi_u, ntype * nwmax * nr_max * sizeof(double));
    hipMemcpy(psi_u, psi_u_now, ntype * nwmax * nr_max * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dpsi_u, dpsi_u_now, ntype * nwmax * nr_max * sizeof(double), hipMemcpyHostToDevice);

    bool *atom_iw2_new;
    int *atom_iw2_ylm;
    hipMalloc((void **)&atom_iw2_new, ntype * nwmax * sizeof(bool));
    hipMalloc((void **)&atom_iw2_ylm, ntype * nwmax * sizeof(int));
    hipMemcpy(atom_iw2_new, atom_iw2_new_now, ntype * nwmax * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(atom_iw2_ylm, atom_iw2_ylm_now, ntype * nwmax * sizeof(int), hipMemcpyHostToDevice);

    int *trace_lo;
    size_t size_trace_lo = NLOCAL_now;
    hipMalloc((void **)&trace_lo, size_trace_lo * sizeof(int));
    hipMemcpy(trace_lo, GridT.trace_lo, size_trace_lo * sizeof(int), hipMemcpyHostToDevice);

    int *itiaiw2iwt;
    hipMalloc((void **)&itiaiw2iwt, size_itiaiw2iwt * sizeof(int));
    hipMemcpy(itiaiw2iwt, itiaiw2iwt_now, size_itiaiw2iwt * sizeof(int), hipMemcpyHostToDevice);

    int *start_ind_g;
    hipMalloc((void **)&start_ind_g, nbxx * sizeof(int));
    hipMemcpy(start_ind_g, start_ind, nbxx * sizeof(int), hipMemcpyHostToDevice);

    double *vldr3;
    hipMalloc((void **)&vldr3, nbz * bxyz * sizeof(double));
    hipMemset(vldr3, 0, nbz * bxyz * sizeof(double));

    double *psir_ylm;
    hipMalloc((void **)&psir_ylm, nbz * max_size * bxyz * nwmax * sizeof(double));
    hipMemset(psir_ylm, 0, nbz * max_size * bxyz * nwmax * sizeof(double));

    bool *cal_flag;
    hipMalloc((void **)&cal_flag, nbz * bxyz * max_size * sizeof(bool));
    hipMemset(cal_flag, 0, nbz * bxyz * max_size * sizeof(bool));

    double *GridVlocal;
    hipMalloc((void **)&GridVlocal, lgd * lgd * sizeof(double));
    hipMemset(GridVlocal, 0, lgd * lgd * sizeof(double));

    const size_t atom_pair_num = (max_size * (max_size + 1)) / 2;

    double *atom_pair_index1 = new double[atom_pair_num];
    double *atom_pair_index2 = new double[atom_pair_num];
    for (int x = 0; x < max_size; x++)
    {
        for (int y = 0; y < max_size - x; y++)
        {
            int index = ((max_size * 2 - x + 1) * x) / 2 + y;
            atom_pair_index1[index] = x;
            atom_pair_index2[index] = y;
        }
    }
    double *atom_pair_index1_g;
    hipMalloc((void **)&atom_pair_index1_g, atom_pair_num * sizeof(double));

    double *atom_pair_index2_g;
    hipMalloc((void **)&atom_pair_index2_g, atom_pair_num * sizeof(double));

    hipMemcpy(atom_pair_index1_g, atom_pair_index1, atom_pair_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(atom_pair_index2_g, atom_pair_index2, atom_pair_num * sizeof(double), hipMemcpyHostToDevice);

    int psi_size = max_size * bxyz;
    const int ALIGN_SIZE = 32;
    int psi_size_up = ((psi_size + ALIGN_SIZE - 1) / ALIGN_SIZE) * ALIGN_SIZE;
    double *dr = new double[psi_size_up * nbz * 4]; // [ x,y,z,distance]
    int *it = new int[psi_size_up * nbz];
    int *psir_ylm_start = new int[psi_size_up * nbz];
    int *num_psir = new int[nbz];
    // begin kernel

    double *dr_g; // [ x,y,z,distance]
    hipMalloc((void **)&dr_g, psi_size_up * nbz * 4 * sizeof(double));

    int *it_g;
    hipMalloc((void **)&it_g, psi_size_up * nbz * sizeof(int));

    int *psir_ylm_start_g;
    hipMalloc((void **)&psir_ylm_start_g, psi_size_up * nbz * sizeof(int));

    int *num_psir_g;
    hipMalloc((void **)&num_psir_g, nbz * sizeof(int));

    hipEventRecord(t2);

    // printf("maxsize=%d\n", max_size);

    for (int i = 0; i < nbx; i++)
    {
        for (int j = 0; j < nby; j++)
        {
            int num_psi_pos = 0;
            for (int z_index = 0; z_index < nbz; z_index++)
            {
                int num_get_psi = 0;
                int grid_index = i * nby * nbz + j * nbz + z_index;
                for (int id = 0; id < GridT.how_many_atoms[grid_index]; id++)
                {
                    for (int ib = 0; ib < bxyz; ib++)
                    {
                        int mcell_index = GridT.bcell_start[grid_index] + id;
                        int imcell = GridT.which_bigcell[mcell_index];
                        int iat = GridT.which_atom[mcell_index];
                        int it_temp = GlobalC::ucell.iat2it[iat];
                        double dr_temp[3];
                        dr_temp[0] = GridT.meshcell_pos[ib][0] + GridT.meshball_positions[imcell][0] - GridT.tau_in_bigcell[iat][0];
                        dr_temp[1] = GridT.meshcell_pos[ib][1] + GridT.meshball_positions[imcell][1] - GridT.tau_in_bigcell[iat][1];
                        dr_temp[2] = GridT.meshcell_pos[ib][2] + GridT.meshball_positions[imcell][2] - GridT.tau_in_bigcell[iat][2];

                        double distance = sqrt(dr_temp[0] * dr_temp[0] + dr_temp[1] * dr_temp[1] + dr_temp[2] * dr_temp[2]);
                        if (distance <= GlobalC::ORB.Phi[it_temp].getRcut())
                        {
                            int pos_temp = num_psi_pos + num_get_psi;
                            if (distance < 1.0E-9) distance += 1.0E-9;
                            dr[pos_temp * 4] = dr_temp[0] / distance;
                            dr[pos_temp * 4 + 1] = dr_temp[1] / distance;
                            dr[pos_temp * 4 + 2] = dr_temp[2] / distance;
                            dr[pos_temp * 4 + 3] = distance;
                            it[pos_temp] = it_temp;
                            int dist_tmp = z_index * bxyz * max_size + ib * max_size + id;
                            psir_ylm_start[pos_temp] = dist_tmp;
                            num_get_psi++;
                        }
                    }
                }
                int num_get_psi_up = ((num_get_psi + ALIGN_SIZE - 1) / ALIGN_SIZE) * ALIGN_SIZE;
                for (;num_get_psi < num_get_psi_up; num_get_psi++)
                {
                    int pos_temp = num_psi_pos + num_get_psi;
                    dr[pos_temp * 4] = 0.0;
                    dr[pos_temp * 4 + 1] = 0.0;
                    dr[pos_temp * 4 + 2] = 0.0;
                    dr[pos_temp * 4 + 3] = 0.0;
                    it[pos_temp] = -1;
                    psir_ylm_start[pos_temp] = psir_ylm_start[pos_temp - 1] + 1;
                }
                num_psir[z_index] = num_get_psi_up; // align to ALIGN_SIZE 32
                num_psi_pos += psi_size_up;
            }

            hipMemcpy(dr_g, dr, psi_size_up * nbz * 4 * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(it_g, it, psi_size_up * nbz * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(psir_ylm_start_g, psir_ylm_start, psi_size_up * nbz * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(num_psir_g, num_psir, nbz * sizeof(int), hipMemcpyHostToDevice);
            hipMemset(cal_flag, 0, nbz * bxyz * max_size * sizeof(bool));

            dim3 grid1(nbz);
            dim3 block1(ALIGN_SIZE); 
            get_psi<<<grid1, block1>>>(dr_g,
                                       it_g,
                                       psir_ylm_start_g,
                                       num_psir_g,
                                       psi_size_up,
                                       cal_flag,
                                       ucell_atom_nwl,
                                       atom_iw2_new,
                                       atom_iw2_ylm,
                                       atom_nw,
                                       nr_max,
                                       psi_u,
                                       dpsi_u,
                                       psir_ylm);
            dim3 grid3(nbz);
            dim3 block3(bx, by, bz);
            get_vldr3<<<grid3, block3>>>(i * nby * nbz + j * nbz,
                                         nbx,
                                         nby,
                                         nbz,
                                         nbz_start,
                                         ncy,
                                         nczp,
                                         vlocal_cu,
                                         start_ind_g,
                                         vldr3);

            dim3 grid4(max_size, max_size, nbz);
            dim3 block4(nwmax, nwmax);
            psi_multiple<<<grid4, block4>>>(i * nby * nbz + j * nbz,
                                            how_many_atoms,
                                            atom_pair_index1_g,
                                            atom_pair_index2_g,
                                            bcell_start,
                                            which_atom,
                                            iat2it,
                                            iat2ia,
                                            itiaiw2iwt,
                                            cal_flag,
                                            psir_ylm,
                                            trace_lo,
                                            atom_nw,
                                            vldr3,
                                            GridVlocal,
                                            lgd);
        } // j
    }     // i

    hipDeviceSynchronize();
    hipMemcpy(GridVlocal_now, GridVlocal, lgd * lgd * sizeof(double), hipMemcpyDeviceToHost);
    // printf("GridVlocal_now[0]: %lf\n", GridVlocal_now[0]);
    hipEventRecord(t3);
    hipDeviceSynchronize();
    // free
    hipFree(vldr3);
    hipFree(psir_ylm);
    hipFree(cal_flag);

    hipFree(how_many_atoms);
    hipFree(bcell_start);
    hipFree(which_bigcell);
    hipFree(which_atom);
    hipFree(iat2it);
    hipFree(iat2ia);
    hipFree(vlocal_cu);
    hipFree(ucell_atom_nwl);
    hipFree(psi_u);
    hipFree(dpsi_u);
    hipFree(atom_iw2_new);
    hipFree(atom_iw2_ylm);
    hipFree(atom_nw);
    hipFree(trace_lo);
    hipFree(itiaiw2iwt);
    hipFree(start_ind_g);
    hipFree(GridVlocal);

    hipFree(atom_pair_index1_g);
    hipFree(atom_pair_index2_g);
    hipFree(dr_g);
    hipFree(it_g);
    hipFree(psir_ylm_start_g);
    hipFree(num_psir_g);

    delete[] atom_pair_index1;
    delete[] atom_pair_index2;
    delete[] dr;
    delete[] it;
    delete[] psir_ylm_start;
    delete[] num_psir;

    delete[] atom_nw_now;
    delete[] itiaiw2iwt_now;
    delete[] ucell_atom_nwl_now;
    delete[] psi_u_now;
    delete[] dpsi_u_now;
    delete[] atom_iw2_new_now;
    delete[] atom_iw2_ylm_now;

    hipEventRecord(t4);
    float copy = 0;
    float calc = 0;
    float free = 0;
    hipEventElapsedTime(&copy, t1, t2);
    hipEventElapsedTime(&calc, t2, t3);
    hipEventElapsedTime(&free, t3, t4);

    printf("copy time = %f\ncal time = %f\nfree time = %f\n", copy, calc, free);
}