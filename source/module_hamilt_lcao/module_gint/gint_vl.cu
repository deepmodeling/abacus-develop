#include "hip/hip_runtime.h"
#include "omp.h"
#include "gint_tools.h"
#include "gint_vl.h"
#include "module_base/ylm.h"
#include "module_basis/module_ao/ORB_read.h"
#include "module_hamilt_pw/hamilt_pwdft/global.h"
#include <fstream>
#include <sstream>
#define __DEBUG
__constant__ double ylmcoef[36];
__constant__ int bx_g[1];
__constant__ int by_g[1];
__constant__ int bz_g[1];
__constant__ int bxyz_g[1];
__constant__ int max_size_g[1];
__constant__ int nwmax_g[1];
__constant__ int namax_g[1];
__constant__ int nnnmax_g[1];
__constant__ int ntype_g[1];
__constant__ double delta_r_g[1];
__constant__ double vfactor_g[1];

void dump_cuda_array_to_file(double *cuda_array, int width, int hight, const std::string &filename)
{
    double *h_data = new double[width * hight];
    hipMemcpy(h_data, cuda_array, width * hight * sizeof(double), hipMemcpyDeviceToHost);

    std::ofstream outFile(filename);
    if (!outFile.is_open())
    {
        std::cerr << "Failed to open file for writing." << std::endl;
    }
    for (int j = 0; j < hight; ++j)
    {
        for (int i = 0; i < width; ++i)
        {
            outFile << "hight" << j << "   width:" << i << "   " << h_data[j * width + i] << std::endl;
        }
    }
    outFile.close();
    delete[] h_data;
}

__global__ void get_psi_and_vldr3(double *dr_all,
                        int *it_all,
                        int *psir_ylm_start,
                        int *ib_index,
                        int *num_psir,
                        int psi_size_max,
                        int *ucell_atom_nwl,
                        bool *atom_iw2_new,
                        int *atom_iw2_ylm,
                        int *atom_nw,
                        int nr_max,
                        double *psi_u,
                        double *dpsi_u,
                        double *psir_ylm_left,
                        double *psir_ylm_right,
                        double *vlocal,
                        int *vindex_local)
{
    int size = num_psir[blockIdx.x];
    {
        extern __shared__ double vldr3[];
        {
            int k = blockIdx.x;
            int start_index = bxyz_g[0] * k;
            int end_index = start_index + bxyz_g[0];
            start_index += threadIdx.x;
            int index = threadIdx.x;
            for (int i = start_index; i < end_index; i += blockDim.x)
            {
                vldr3[index] = vlocal[vindex_local[i]] * vfactor_g[0];
                index += blockDim.x;
            }
        }
        __syncthreads();
        {
            int start_index = psi_size_max * blockIdx.x;
            int end_index = start_index + size;
            start_index += threadIdx.x;
            for (int index = start_index; index < end_index; index += blockDim.x)
            {
                int it = it_all[index];
                if (it < 0)
                    continue;

                double dr[3];
                dr[0] = dr_all[index * 4];
                dr[1] = dr_all[index * 4 + 1];
                dr[2] = dr_all[index * 4 + 2];
                double distance = dr_all[index * 4 + 3];

                // begin calculation
                double ylma[150];
                /***************************
                L = 0
                ***************************/
                ylma[0] = ylmcoef[0]; // l=0, m=0
                int nwl = ucell_atom_nwl[it];

                if (nwl == 0)
                    goto YLM_END;

                /***************************
                L = 1
                ***************************/
                ylma[1] = ylmcoef[1] * dr[2];  // l=1, m=0
                ylma[2] = -ylmcoef[1] * dr[0]; // l=1, m=1
                ylma[3] = -ylmcoef[1] * dr[1]; // l=1, m=-1
                if (nwl == 1)
                    goto YLM_END;

                /***************************
                L = 2
                ***************************/
                ylma[4] = ylmcoef[2] * dr[2] * ylma[1] - ylmcoef[3] * ylma[0]; // l=2, m=0
                {
                    double tmp0 = ylmcoef[4] * dr[2];
                    ylma[5] = tmp0 * ylma[2]; // l=2,m=1
                    ylma[6] = tmp0 * ylma[3]; // l=2,m=-1

                    tmp0 = ylmcoef[4] * dr[0];
                    ylma[7] = ylmcoef[5] * ylma[4] - ylmcoef[6] * ylma[0] - tmp0 * ylma[2]; // l=2,m=2
                    ylma[8] = -tmp0 * ylma[3];
                    //	ylma[grid_index*nnnmax+8] = tmp1+tmp2*ylma[grid_index*nnnmax+3];//l=2,m=-2
                    if (nwl == 2)
                        goto YLM_END;

                    /***************************
                    L = 3
                    ***************************/
                    ylma[9] = ylmcoef[7] * dr[2] * ylma[4] - ylmcoef[8] * ylma[1]; // l=3, m=0

                    tmp0 = ylmcoef[9] * dr[2];
                    ylma[10] = tmp0 * ylma[5] - ylmcoef[10] * ylma[2]; // l=3,m=1
                    ylma[11] = tmp0 * ylma[6] - ylmcoef[10] * ylma[3]; // l=3,m=-1

                    tmp0 = ylmcoef[11] * dr[2];
                    ylma[12] = tmp0 * ylma[7]; // l=3,m=2
                    ylma[13] = tmp0 * ylma[8]; // l=3,m=-2

                    tmp0 = ylmcoef[14] * dr[0];
                    ylma[14] = ylmcoef[12] * ylma[10] - ylmcoef[13] * ylma[2] - tmp0 * ylma[7]; // l=3,m=3
                    ylma[15] = ylmcoef[12] * ylma[11] - ylmcoef[13] * ylma[3] - tmp0 * ylma[8]; // l=3,m=-3
                    if (nwl == 3)
                        goto YLM_END;

                    /***************************
                    L = 4
                    ***************************/
                    ylma[16] = ylmcoef[15] * dr[2] * ylma[9] - ylmcoef[16] * ylma[4]; // l=4,m=0

                    tmp0 = ylmcoef[17] * dr[2];
                    ylma[17] = tmp0 * ylma[10] - ylmcoef[18] * ylma[5]; // l=4,m=1
                    ylma[18] = tmp0 * ylma[11] - ylmcoef[18] * ylma[6]; // l=4,m=-1

                    tmp0 = ylmcoef[19] * dr[2];
                    ylma[19] = tmp0 * ylma[12] - ylmcoef[20] * ylma[7]; // l=4,m=2
                    ylma[20] = tmp0 * ylma[13] - ylmcoef[20] * ylma[8]; // l=4,m=-2

                    tmp0 = 3.0 * dr[2];
                    ylma[21] = tmp0 * ylma[14]; // l=4,m=3
                    ylma[22] = tmp0 * ylma[15]; // l=4,m=-3

                    tmp0 = ylmcoef[23] * dr[0];
                    ylma[23] = ylmcoef[21] * ylma[19] - ylmcoef[22] * ylma[7] - tmp0 * ylma[14]; // l=4,m=4
                    ylma[24] = ylmcoef[21] * ylma[20] - ylmcoef[22] * ylma[8] - tmp0 * ylma[15]; // l=4,m=-4
                    if (nwl == 4)
                        goto YLM_END;

                    /***************************
                    L = 5
                    ***************************/
                    ylma[25] = ylmcoef[24] * dr[2] * ylma[16] - ylmcoef[25] * ylma[9]; // l=5,m=0

                    tmp0 = ylmcoef[26] * dr[2];
                    ylma[26] = tmp0 * ylma[17] - ylmcoef[27] * ylma[10]; // l=5,m=1
                    ylma[27] = tmp0 * ylma[18] - ylmcoef[27] * ylma[11]; // l=5,m=-1

                    tmp0 = ylmcoef[28] * dr[2];
                    ylma[28] = tmp0 * ylma[19] - ylmcoef[29] * ylma[12]; // l=5,m=2
                    ylma[29] = tmp0 * ylma[20] - ylmcoef[29] * ylma[13]; // l=5,m=-2

                    tmp0 = ylmcoef[30] * dr[2];
                    ylma[30] = tmp0 * ylma[21] - ylmcoef[31] * ylma[14]; // l=5,m=3
                    ylma[31] = tmp0 * ylma[22] - ylmcoef[31] * ylma[15]; // l=5,m=-3

                    tmp0 = ylmcoef[32] * dr[2];
                    ylma[32] = tmp0 * ylma[23]; // l=5,m=4
                    ylma[33] = tmp0 * ylma[24]; // l=5,m=-4

                    tmp0 = ylmcoef[35] * dr[0];
                    ylma[34] = ylmcoef[33] * ylma[30] - ylmcoef[34] * ylma[14] - tmp0 * ylma[23]; // l=5,m=5
                    ylma[35] = ylmcoef[33] * ylma[31] - ylmcoef[34] * ylma[15] - tmp0 * ylma[24]; // l=5,m=-5
                    if (nwl == 5)
                        goto YLM_END;

                    // if nwl > 5
                    for (int il = 6; il <= nwl; il++)
                    {
                        int istart = il * il;
                        int istart1 = (il - 1) * (il - 1);
                        int istart2 = (il - 2) * (il - 2);

                        double fac2 = sqrt(4.0 * istart - 1.0);
                        double fac4 = sqrt(4.0 * istart1 - 1.0);

                        for (int im = 0; im < 2 * il - 1; im++)
                        {
                            int imm = (im + 1) / 2;
                            //			if (im % 2 == 0) imm *= -1;

                            ylma[istart + im] = fac2 / sqrt((double)istart - imm * imm) * (dr[2] * ylma[istart1 + im] - sqrt((double)istart1 - imm * imm) / fac4 * ylma[istart2 + im]);
                        }

                        double bl1 = sqrt(2.0 * il / (2.0 * il + 1.0));
                        double bl2 = sqrt((2.0 * il - 2.0) / (2.0 * il - 1.0));
                        double bl3 = sqrt(2.0) / fac2;

                        ylma[istart + 2 * il - 1] = (bl3 * ylma[istart + 2 * il - 5] - bl2 * ylma[istart2 + 2 * il - 5] - 2.0 * dr[0] * ylma[istart1 + 2 * il - 3]) / bl1;
                        ylma[istart + 2 * il] = (bl3 * ylma[istart + 2 * il - 4] - bl2 * ylma[istart2 + 2 * il - 4] - 2.0 * dr[0] * ylma[istart1 + 2 * il - 2]) / bl1;
                    }
                }
            YLM_END:
                distance /= delta_r_g[0];

                int ip = (int)(distance);
                double dx = distance - ip;
                double dx2 = dx * dx;
                double dx3 = dx2 * dx;

                double c3 = 3.0 * dx2 - 2.0 * dx3;
                double c1 = 1.0 - c3;
                double c2 = (dx - 2.0 * dx2 + dx3) * delta_r_g[0];
                double c4 = (dx3 - dx2) * delta_r_g[0];

                double phi = 0.0;
                int it_nw = it * nwmax_g[0];
                const int it_nw_nr_ip = it_nw * nr_max + ip;
                int iw_nr = it_nw_nr_ip;
                int dist_tmp = psir_ylm_start[index] * nwmax_g[0];
                double vlbr3_value = vldr3[ib_index[index]];
                int it_nw_iw = it_nw;
                for (int iw = 0; iw < atom_nw[it]; ++iw)
                {
                    if (atom_iw2_new[it_nw_iw])
                    {
                        phi = c1 * psi_u[iw_nr] + c2 * dpsi_u[iw_nr] + c3 * psi_u[iw_nr + 1] + c4 * dpsi_u[iw_nr + 1];
                    }
                    double temp = phi * ylma[atom_iw2_ylm[it_nw_iw]];
                    psir_ylm_right[dist_tmp] = temp;
                    psir_ylm_left[dist_tmp] = temp * vlbr3_value;
                    dist_tmp++;
                    iw_nr += nr_max;
                    it_nw_iw++;
                }
            }
        }
    } // if size
}

__global__ void psi_multiple(int *atom_pair_input_info_g,
                             int *num_atom_pair_g,
                             int grid_index,
                             double *psir_ylm_left,
                             double *psir_ylm_right,
                             int atom_pair_size_of_meshcell,
                             double *GridVlocal,
                             int lgd)
{
    //int k = blockIdx.x;
    grid_index += blockIdx.x;
    int atom_pair_num = num_atom_pair_g[blockIdx.x];
    int start_index = atom_pair_size_of_meshcell * blockIdx.x;
    int end_index = start_index + atom_pair_num;
    start_index += blockIdx.y * 6;
    int step = gridDim.y * 6;
    for (int atom_pair_index = start_index; atom_pair_index < end_index; atom_pair_index += step)
    {
        int atomnow1 = atom_pair_input_info_g[atom_pair_index];
        int atomnow2 = atom_pair_input_info_g[atom_pair_index + 1];
        int iw1 = threadIdx.x;
        int iw2 = threadIdx.y;
        if (iw1 >= atom_pair_input_info_g[atom_pair_index + 2] || iw2 >= atom_pair_input_info_g[atom_pair_index + 3])
        {
            return;
        }
        int lo1_iw1 = atom_pair_input_info_g[atom_pair_index + 4] + iw1;
        int lo2_iw2 = atom_pair_input_info_g[atom_pair_index + 5] + iw2;
        double v2 = 0.0;
        int vldr3_index = blockIdx.x * bxyz_g[0];

        for (int ib = 0; ib < bxyz_g[0]; ++ib)
        {
            int calc_index1 = vldr3_index * max_size_g[0];
            int calc_index2 = calc_index1 + atomnow2;
            calc_index1 += atomnow1;
            v2 += psir_ylm_left[calc_index1 * nwmax_g[0] + iw1] * psir_ylm_right[calc_index2 * nwmax_g[0] + iw2];
            vldr3_index++;
        }
        atomicAdd(&(GridVlocal[lo1_iw1 * lgd + lo2_iw2]), v2);
    }
}

void gint_gamma_vl_gpu(hamilt::HContainer<double>* hRGint,
                       const int lgd,
                       const int nnnmax,
                       const int max_size,
                       const double vfactor,
                       const double *vlocal,
                       const double *ylmcoef_now,
                       const int bx,
                       const int by,
                       const int bz,
                       const int bxyz,
                       const int ncx,
                       const int ncy,
                       const int nczp,
                       const int NLOCAL_now,
                       const int nbxx,
                       int *start_ind,
                       const Grid_Technique &GridT)
{
    // printf("\n**************START GPU SEG***************\n");
#ifdef __DEBUG
    hipEvent_t t1, t2, t3, t4;
    hipEventCreate(&t1);
    hipEventCreate(&t2);
    hipEventCreate(&t3);
    hipEventCreate(&t4);

    hipEventRecord(t1);
#endif

    const Numerical_Orbital_Lm *pointer;
    // const double delta_r = GlobalC::ORB.dr_uniform;
    // const int total_atoms_on_grid = GridT.total_atoms_on_grid;
    const int nbx = GridT.nbx;
    const int nby = GridT.nby;
    const int nbz = GridT.nbzp;
    const int nwmax = GlobalC::ucell.nwmax;
    const int namax = GlobalC::ucell.namax;
    const int ntype = GlobalC::ucell.ntype;

    double max_cut = 0;
    for (int i = 0; i < ntype; i++)
    {
        if (GlobalC::ORB.Phi[i].getRcut() > max_cut)
        {
            max_cut = GlobalC::ORB.Phi[i].getRcut();
        }
    }

    int *atom_nw_now = new int[ntype];
    int *ucell_atom_nwl_now = new int[ntype];
    for (int i = 0; i < ntype; i++)
    {
        atom_nw_now[i] = GlobalC::ucell.atoms[i].nw;
        ucell_atom_nwl_now[i] = GlobalC::ucell.atoms[i].nwl;
    }

    int nr_max = static_cast<int>(1000 * max_cut) + 10;
    double *psi_u_now = new double[ntype * nwmax * nr_max];
    double *dpsi_u_now = new double[ntype * nwmax * nr_max];
    bool *atom_iw2_new_now = new bool[ntype * nwmax];
    int *atom_iw2_ylm_now = new int[ntype * nwmax];

    Atom *atomx;
    for (int i = 0; i < ntype; i++)
    {
        atomx = &GlobalC::ucell.atoms[i];
        for (int j = 0; j < nwmax; j++)
        {
            if (j < atomx->nw)
            {
                atom_iw2_new_now[i * nwmax + j] = atomx->iw2_new[j];
                atom_iw2_ylm_now[i * nwmax + j] = atomx->iw2_ylm[j];
                pointer = &GlobalC::ORB.Phi[i].PhiLN(atomx->iw2l[j], atomx->iw2n[j]);
                for (int k = 0; k < nr_max; k++)
                {
                    if (k < pointer->nr_uniform)
                    {
                        psi_u_now[i * nwmax * nr_max + j * nr_max + k] = pointer->psi_uniform[k];
                        dpsi_u_now[i * nwmax * nr_max + j * nr_max + k] = pointer->dpsi_uniform[k];
                    }
                    else
                    {
                        psi_u_now[i * nwmax * nr_max + j * nr_max + k] = 0;
                        dpsi_u_now[i * nwmax * nr_max + j * nr_max + k] = 0;
                    }
                }
            }
            else
            {
                atom_iw2_new_now[i * nwmax + j] = false;
                atom_iw2_ylm_now[i * nwmax + j] = 0;
                for (int k = 0; k < nr_max; k++)
                {
                    psi_u_now[i * nwmax * nr_max + j * nr_max + k] = 0;
                    dpsi_u_now[i * nwmax * nr_max + j * nr_max + k] = 0;
                }
            }
        }
    }

    hipMemcpyToSymbol(HIP_SYMBOL(ylmcoef), ylmcoef_now, 36 * sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(bx_g), &bx, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(by_g), &by, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(bz_g), &bz, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(bxyz_g), &bxyz, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(max_size_g), &max_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(nwmax_g), &nwmax, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(namax_g), &namax, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(nnnmax_g), &nnnmax, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(ntype_g), &ntype, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(delta_r_g), &GlobalC::ORB.dr_uniform, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(vfactor_g), &vfactor, sizeof(double));

    // read only
    double *vlocal_cu;
    hipMalloc((void **)&vlocal_cu, ncx * ncy * nczp * sizeof(double));
    hipMemcpy(vlocal_cu, vlocal, ncx * ncy * nczp * sizeof(double), hipMemcpyHostToDevice);

    int *atom_nw_g;
    hipMalloc((void **)&atom_nw_g, ntype * sizeof(int));
    hipMemcpy(atom_nw_g, atom_nw_now, ntype * sizeof(int), hipMemcpyHostToDevice);

    int *ucell_atom_nwl;
    hipMalloc((void **)&ucell_atom_nwl, ntype * sizeof(int));
    hipMemcpy(ucell_atom_nwl, ucell_atom_nwl_now, ntype * sizeof(int), hipMemcpyHostToDevice);

    double *psi_u; //TODO  合并这两个数组
    double *dpsi_u;
    hipMalloc((void **)&psi_u, ntype * nwmax * nr_max * sizeof(double));
    hipMalloc((void **)&dpsi_u, ntype * nwmax * nr_max * sizeof(double));
    hipMemcpy(psi_u, psi_u_now, ntype * nwmax * nr_max * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dpsi_u, dpsi_u_now, ntype * nwmax * nr_max * sizeof(double), hipMemcpyHostToDevice);

    bool *atom_iw2_new;
    int *atom_iw2_ylm;
    hipMalloc((void **)&atom_iw2_new, ntype * nwmax * sizeof(bool));
    hipMalloc((void **)&atom_iw2_ylm, ntype * nwmax * sizeof(int));
    hipMemcpy(atom_iw2_new, atom_iw2_new_now, ntype * nwmax * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(atom_iw2_ylm, atom_iw2_ylm_now, ntype * nwmax * sizeof(int), hipMemcpyHostToDevice);

    double *vldr3;
    hipMalloc((void **)&vldr3, nbz * bxyz * sizeof(double));
    hipMemset(vldr3, 0, nbz * bxyz * sizeof(double));

    double *psir_ylm_left;
    hipMalloc((void **)&psir_ylm_left, nbz * max_size * bxyz * nwmax * sizeof(double));
    hipMemset(psir_ylm_left, 0, nbz * max_size * bxyz * nwmax * sizeof(double));

    double *psir_ylm_right;
    hipMalloc((void **)&psir_ylm_right, nbz * max_size * bxyz * nwmax * sizeof(double));
    hipMemset(psir_ylm_right, 0, nbz * max_size * bxyz * nwmax * sizeof(double));

    double *GridVlocal_now = new double[lgd * lgd];

    double *GridVlocal;
    hipMalloc((void **)&GridVlocal, lgd * lgd * sizeof(double));
    hipMemset(GridVlocal, 0, lgd * lgd * sizeof(double));

    const int ALIGN_SIZE = 32;

    const int atom_pair_size_of_meshcell = max_size * max_size * 6;

    const int atom_pair_size_over_nbz = atom_pair_size_of_meshcell * nbz;

    int *num_atom_pair = new int[nbz];
    int *atom_pair_input_info = new int[atom_pair_size_over_nbz];

    int *atom_pair_input_info_g;
    hipMalloc((void **)&atom_pair_input_info_g, atom_pair_size_over_nbz * sizeof(int));

    int *num_atom_pair_g;
    hipMalloc((void **)&num_atom_pair_g, nbz * sizeof(int));

    int psi_size_max = max_size * bxyz;

    double *dr = new double[psi_size_max * nbz * 4]; // [ x,y,z,distance]
    // TODO 合并下面这三个
    int *it = new int[psi_size_max * nbz];
    int *psir_ylm_start = new int[psi_size_max * nbz];
    int *ib_index = new int[psi_size_max * nbz];
    int *num_psir = new int[nbz];

    int *vindex = new int[nbz * bxyz];
    // begin kernel
    
    double *dr_g; // [ x,y,z,distance]
    hipMalloc((void **)&dr_g, psi_size_max * nbz * 4 * sizeof(double));

    int *it_g;
    hipMalloc((void **)&it_g, psi_size_max * nbz * sizeof(int));

    int *psir_ylm_start_g;
    hipMalloc((void **)&psir_ylm_start_g, psi_size_max * nbz * sizeof(int));

    int *ib_index_g;
    hipMalloc((void **)&ib_index_g, psi_size_max * nbz * sizeof(int));

    int *num_psir_g;
    hipMalloc((void **)&num_psir_g, nbz * sizeof(int));

    int *vindex_g;
    hipMalloc((void **)&vindex_g, nbz * bxyz * sizeof(int));

#ifdef __DEBUG

    hipEventRecord(t2);

    // printf("maxsize=%d\n", max_size);
    float copy_per_calc = 0;
    float calc_psi = 0;
    float calc_multiple = 0;
#endif
    for (int i = 0; i < nbx; i++)
    {
        for (int j = 0; j < nby; j++)
        {
#ifdef __DEBUG
            hipEvent_t t1_5, t1_6, t1_7, t1_8;
            hipEventCreate(&t1_5);
            hipEventCreate(&t1_6);
            hipEventCreate(&t1_7);
            hipEventCreate(&t1_8);

            hipEventRecord(t1_5);
#endif
            for (int z_index = 0; z_index < nbz; z_index++)
            {
                int num_get_psi = 0;
                int grid_index = i * nby * nbz + j * nbz + z_index;
                int num_psi_pos = psi_size_max * z_index;
                for (int id = 0; id < GridT.how_many_atoms[grid_index]; id++)
                {
                    for (int ib = 0; ib < bxyz; ib++)
                    {
                        int mcell_index = GridT.bcell_start[grid_index] + id;
                        int imcell = GridT.which_bigcell[mcell_index];
                        int iat = GridT.which_atom[mcell_index];
                        int it_temp = GlobalC::ucell.iat2it[iat];
                        double dr_temp[3];
                        dr_temp[0] = GridT.meshcell_pos[ib][0] + GridT.meshball_positions[imcell][0] - GridT.tau_in_bigcell[iat][0];
                        dr_temp[1] = GridT.meshcell_pos[ib][1] + GridT.meshball_positions[imcell][1] - GridT.tau_in_bigcell[iat][1];
                        dr_temp[2] = GridT.meshcell_pos[ib][2] + GridT.meshball_positions[imcell][2] - GridT.tau_in_bigcell[iat][2];

                        double distance = sqrt(dr_temp[0] * dr_temp[0] + dr_temp[1] * dr_temp[1] + dr_temp[2] * dr_temp[2]);
                        if (distance <= GlobalC::ORB.Phi[it_temp].getRcut())
                        {
                            int pos_temp = num_psi_pos + num_get_psi;
                            if (distance < 1.0E-9)
                                distance += 1.0E-9;
                            dr[pos_temp * 4] = dr_temp[0] / distance;
                            dr[pos_temp * 4 + 1] = dr_temp[1] / distance;
                            dr[pos_temp * 4 + 2] = dr_temp[2] / distance;
                            dr[pos_temp * 4 + 3] = distance;
                            it[pos_temp] = it_temp;
                            int dist_tmp = z_index * bxyz * max_size + ib * max_size + id;
                            psir_ylm_start[pos_temp] = dist_tmp;
                            ib_index[pos_temp] = ib;
                            num_get_psi++;
                        }
                    }
                }
                num_psir[z_index] = num_get_psi;

                int vindex_temp = z_index * bxyz;
                for (int bx_index = 0; bx_index < bx; bx_index++)
                {
                    for (int by_index = 0; by_index < by; by_index++)
                    {
                        for (int bz_index = 0; bz_index < bz; bz_index++)
                        {
                            int vindex_global = bx_index * ncy * nczp + by_index * nczp + bz_index + start_ind[grid_index];
                            vindex[vindex_temp] = vindex_global;
                            vindex_temp++;
                        }
                    }
                }

                int atom_pair_index_in_nbz = atom_pair_size_of_meshcell * z_index;
                int atom_pair_index_in_meshcell = 0;
                for (int atom1 = 0; atom1 < GridT.how_many_atoms[grid_index]; atom1++)
                {
                    for (int atom2 = 0; atom2 < GridT.how_many_atoms[grid_index]; atom2++)
                    {
                        int iat1 = GridT.which_atom[GridT.bcell_start[grid_index] + atom1];
                        int iat2 = GridT.which_atom[GridT.bcell_start[grid_index] + atom2];
                        int it1 = GlobalC::ucell.iat2it[iat1];
                        int it2 = GlobalC::ucell.iat2it[iat2];
                        int lo1 = GridT.trace_lo[GlobalC::ucell.itiaiw2iwt(it1, GlobalC::ucell.iat2ia[iat1], 0)];
                        int lo2 = GridT.trace_lo[GlobalC::ucell.itiaiw2iwt(it2, GlobalC::ucell.iat2ia[iat2], 0)];
                        if (lo1 <= lo2)
                        {
                            int atom_pair_index = atom_pair_index_in_nbz + atom_pair_index_in_meshcell;
                            atom_pair_input_info[atom_pair_index] = atom1;
                            atom_pair_input_info[atom_pair_index + 1] = atom2;
                            atom_pair_input_info[atom_pair_index + 2] = atom_nw_now[it1];
                            atom_pair_input_info[atom_pair_index + 3] = atom_nw_now[it2];
                            atom_pair_input_info[atom_pair_index + 4] = lo1;
                            atom_pair_input_info[atom_pair_index + 5] = lo2;
                            atom_pair_index_in_meshcell += 6;
                        }
                    }
                }
                num_atom_pair[z_index] = atom_pair_index_in_meshcell;
            }
            hipMemcpy(dr_g, dr, psi_size_max * nbz * 4 * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(it_g, it, psi_size_max * nbz * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(psir_ylm_start_g, psir_ylm_start, psi_size_max * nbz * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(ib_index_g, ib_index, psi_size_max * nbz * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(num_psir_g, num_psir, nbz * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(vindex_g, vindex, nbz * bxyz * sizeof(int), hipMemcpyHostToDevice);
            hipMemset(psir_ylm_left, 0, nbz * max_size * bxyz * nwmax * sizeof(double));
            hipMemset(psir_ylm_right, 0, nbz * max_size * bxyz * nwmax * sizeof(double));

            hipMemcpy(atom_pair_input_info_g, atom_pair_input_info, atom_pair_size_over_nbz * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(num_atom_pair_g, num_atom_pair, nbz * sizeof(int), hipMemcpyHostToDevice);
#ifdef __DEBUG
            hipEventRecord(t1_6);
            hipDeviceSynchronize();
#endif
            dim3 grid1(nbz);
            dim3 block1(ALIGN_SIZE);
            int shared_size = bxyz;
            get_psi_and_vldr3<<<grid1, block1, shared_size>>>(dr_g,
                                       it_g,
                                       psir_ylm_start_g,
                                       ib_index_g,
                                       num_psir_g,
                                       psi_size_max,
                                       ucell_atom_nwl,
                                       atom_iw2_new,
                                       atom_iw2_ylm,
                                       atom_nw_g,
                                       nr_max,
                                       psi_u,
                                       dpsi_u,
                                       psir_ylm_left,
                                       psir_ylm_right,
                                       vlocal_cu,
                                       vindex_g);
#ifdef __DEBUG
            hipDeviceSynchronize();
            hipEventRecord(t1_7);
            hipDeviceSynchronize();
#endif
            dim3 grid4(nbz, 256);
            dim3 block4(nwmax, nwmax);
            psi_multiple<<<grid4, block4>>>(atom_pair_input_info_g,
                                            num_atom_pair_g,
                                            i * nby * nbz + j * nbz,
                                            psir_ylm_left,
                                            psir_ylm_right,
                                            atom_pair_size_of_meshcell,
                                            GridVlocal,
                                            lgd);

#ifdef __DEBUG
            hipDeviceSynchronize();
            hipEventRecord(t1_8);
            float copy_per_calc_temp = 0;
            float calc_psi_temp = 0;
            float calc_multiple_temp = 0;
            hipDeviceSynchronize();

            hipEventElapsedTime(&copy_per_calc_temp, t1_5, t1_6);
            hipEventElapsedTime(&calc_psi_temp, t1_6, t1_7);
            hipEventElapsedTime(&calc_multiple_temp, t1_7, t1_8);
            copy_per_calc += copy_per_calc_temp;
            calc_psi += calc_psi_temp;
            calc_multiple += calc_multiple_temp;
#endif

        } // j
    }     // i

    hipMemcpy(GridVlocal_now, GridVlocal, lgd * lgd * sizeof(double), hipMemcpyDeviceToHost);

    for (int iat1 = 0; iat1 < GlobalC::ucell.nat; iat1++)
    {
        for (int iat2 = 0; iat2 < GlobalC::ucell.nat; iat2++)
        {
            int it1 = GlobalC::ucell.iat2it[iat1];
            int it2 = GlobalC::ucell.iat2it[iat2];
            int lo1 = GridT.trace_lo[GlobalC::ucell.itiaiw2iwt(it1, GlobalC::ucell.iat2ia[iat1], 0)];
            int lo2 = GridT.trace_lo[GlobalC::ucell.itiaiw2iwt(it2, GlobalC::ucell.iat2ia[iat2], 0)];
            if (lo1 <= lo2)
            {
				hamilt::AtomPair<double>* tmp_ap = hRGint->find_pair(iat1, iat2);
                int orb_index = 0;
                if (tmp_ap == NULL) continue;
                for(int orb_i = 0; orb_i < tmp_ap->get_row_size();orb_i++)
                {
                    for(int orb_j = 0;orb_j < tmp_ap->get_col_size();orb_j++)
                    {
                        tmp_ap->get_pointer(0)[orb_index] = GridVlocal_now[(lo1 + orb_i) * lgd + (lo2 + orb_j)];
                        orb_index++;
                    }
                }
            }
        }
    }
#ifdef __DEBUG

    // printf("GridVlocal_now[0]: %lf\n", GridVlocal_now[0]);
    hipEventRecord(t3);
#endif

    // free
    hipFree(vldr3);
    hipFree(psir_ylm_left);
    hipFree(psir_ylm_right);
    hipFree(atom_nw_g);

    hipFree(vlocal_cu);
    hipFree(ucell_atom_nwl);
    hipFree(psi_u);
    hipFree(dpsi_u);
    hipFree(atom_iw2_new);
    hipFree(atom_iw2_ylm);
    hipFree(GridVlocal);

    hipFree(atom_pair_input_info_g);
    hipFree(num_atom_pair_g);

    hipFree(dr_g);
    hipFree(it_g);
    hipFree(psir_ylm_start_g);
    hipFree(ib_index_g);
    hipFree(num_psir_g);
    hipFree(vindex_g);

    delete[] atom_pair_input_info;
    delete[] num_atom_pair;

    delete[] dr;
    delete[] it;
    delete[] psir_ylm_start;
    delete[] ib_index;
    delete[] num_psir;
    delete[] vindex;

    delete[] GridVlocal_now;
    delete[] atom_nw_now;
    delete[] ucell_atom_nwl_now;
    delete[] psi_u_now;
    delete[] dpsi_u_now;
    delete[] atom_iw2_new_now;
    delete[] atom_iw2_ylm_now;

#ifdef __DEBUG
    hipEventRecord(t4);
    float copy = 0;
    float calc = 0;
    float free = 0;
    hipEventElapsedTime(&copy, t1, t2);
    hipEventElapsedTime(&calc, t2, t3);
    hipEventElapsedTime(&free, t3, t4);

    printf("copy time = %f\ncal time = %f\nfree time = %f\n", copy, calc, free);
    printf("copy_per calc time = %f\ncal psi time = %f\nmultiple time = %f\n", copy_per_calc, calc_psi, calc_multiple);
#endif

}