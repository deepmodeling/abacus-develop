#include "module_hamilt_lcao/module_gint/gint_gpu/vbatch_matrix_multiple/cuda_tools.cuh"
#include <iostream>
hipError_t checkCuda(hipError_t result)
{
#if defined(__DEBUG)
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
#endif
    return result;
}

hipError_t checkCudaLastError()
{
#if defined(__DEBUG)
    hipError_t result = hipGetLastError();
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
#else
    return hipSuccess;
#endif
}

void dump_cuda_array_to_file(double *cuda_array, int width, int hight, const std::string &filename)
{
    double *h_data = new double[width * hight];
    hipMemcpy(h_data, cuda_array, width * hight * sizeof(double), hipMemcpyDeviceToHost);

    // std::ofstream outFile(filename);
    // if (!outFile.is_open())
    // {
    //     std::cerr << "Failed to open file for writing." << std::endl;
    // }
    for (int j = 0; j < hight; ++j)
    {
        for (int i = 0; i < width; ++i)
        {
            // outFile << "hight" << j << "   width:" << i << "   " << h_data[j * width + i] << std::endl;
            std::cout << j * width + i << "  " << h_data[j * width + i] << std::endl;
        }
    }
    // outFile.close();
    delete[] h_data;
}

template <typename T>
Cuda_Mem_Wrapper<T>::Cuda_Mem_Wrapper(int one_stream_size, int stream_number, bool malloc_host)
{
    this->stream_number = stream_number;
    this->one_stream_size = one_stream_size;
    this->total_size = stream_number * one_stream_size;
    checkCuda(hipMalloc((void **)&this->device_pointer, this->total_size * sizeof(T)));
    checkCuda(hipMemset(this->device_pointer, 0, this->total_size * sizeof(T)));
    this->host_pointer = nullptr;

    if (malloc_host)
    {
        checkCuda(hipHostMalloc((void **)&this->host_pointer, this->total_size * sizeof(T)));
        memset(this->host_pointer, 0, this->total_size * sizeof(T));
    }
}

template <typename T>
void Cuda_Mem_Wrapper<T>::free_all()
{
    checkCuda(hipFree(this->device_pointer));
    if (this->host_pointer != nullptr)
    {
        checkCuda(hipHostFree(this->host_pointer));
    }
}

template <typename T>
Cuda_Mem_Wrapper<T>::~Cuda_Mem_Wrapper()
{
    this->free_all();
}

template <typename T>
void Cuda_Mem_Wrapper<T>::copy_host_to_device_sync(int stream_id)
{
    #if defined(__DEBUG)
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy host to device" << std::endl;
        exit(1);
    }
    #endif
    checkCuda(hipMemcpy(this->device_pointer + stream_id * this->one_stream_size, this->host_pointer + stream_id * this->one_stream_size,
               this->one_stream_size * sizeof(T), hipMemcpyHostToDevice));
}

template <typename T>
void Cuda_Mem_Wrapper<T>::copy_host_to_device_async(int stream_id, hipStream_t stream)
{
    #if defined(__DEBUG)
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy host to device" << std::endl;
        exit(1);
    }
    #endif
    checkCuda(hipMemcpyAsync(this->device_pointer + stream_id * this->one_stream_size,
                              this->host_pointer + stream_id * this->one_stream_size,
                              this->one_stream_size * sizeof(T), hipMemcpyHostToDevice, stream));
}

template <typename T>
void Cuda_Mem_Wrapper<T>::copy_device_to_host_sync(int stream_id)
{
    #if defined(__DEBUG)
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy device to host" << std::endl;
        exit(1);
    }
    #endif
    checkCuda(hipMemcpy(this->host_pointer + stream_id * this->one_stream_size, this->device_pointer + stream_id * this->one_stream_size,
               this->one_stream_size * sizeof(T), hipMemcpyDeviceToHost));
}

template <typename T>
void Cuda_Mem_Wrapper<T>::copy_device_to_host_async(int stream_id, hipStream_t stream)
{
    #if defined(__DEBUG)
    if (this->host_pointer == nullptr || this->device_pointer == nullptr)
    {
        std::cerr << "host_pointer is nullptr, can not copy device to host" << std::endl;
        exit(1);
    }
    #endif
    checkCuda(hipMemcpyAsync(this->host_pointer + stream_id * this->one_stream_size,
                              this->device_pointer + stream_id * this->one_stream_size,
                              this->one_stream_size * sizeof(T), hipMemcpyDeviceToHost, stream));
}

template <typename T>
T* Cuda_Mem_Wrapper<T>::get_device_pointer(int stream_id)
{
    return this->device_pointer + stream_id * this->one_stream_size;
}

template <typename T>
T* Cuda_Mem_Wrapper<T>::get_host_pointer(int stream_id)
{
    return this->host_pointer + stream_id * this->one_stream_size;
}
template class Cuda_Mem_Wrapper<double>;
template class Cuda_Mem_Wrapper<double*>;
template class Cuda_Mem_Wrapper<int>;