#include "hip/hip_runtime.h"
#include <functional>
#include "vbatch_matrix_mul.cuh"
#include "cuda_tools.cuh"
#include "module_base/blas_connector.h"
#include "module_hamilt_pw/hamilt_pwdft/global.h"

#define sA(i,j)    sA[(j)*slda + (i)]
#define sB(i,j)    sB[(j)*sldb + (i)]
#define fetch(A, m, n, bound)  offs_d##A[min(n*LD##A+m, bound)]
hipError_t checkCuda(hipError_t result)
{
#if defined(__DEBUG)
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
#endif
    return result;
}

hipError_t checkCudaLastError()
{
#if defined(__DEBUG)
    hipError_t result = hipGetLastError();
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
#else
    return hipSuccess;
#endif
}


template<typename T, int DIM_X, int DIM_Y, int BLK_M, int BLK_N, int BLK_K,
         int DIM_XA, int DIM_YA, int DIM_XB, int DIM_YB,
         int THR_M, int THR_N>
static __device__
void vbatched_gemm_device(
    int M, int N, int K,
    T* __restrict__ A, int LDA,
    T* __restrict__ B, int LDB,
    T*       __restrict__ C, int LDC,
    T* sA, int slda,
    T* sB, int sldb)
{
    int idx = threadIdx.x;  // thread's m dimension
    int idy = threadIdx.y;  // thread's n dimension

    int idt = DIM_X * idy + idx;    // thread's global number

    int idxA = idt % DIM_XA;    // idx within A
    int idyA = idt / DIM_XA;    // idy within A

    int idxB = idt % DIM_XB;    // idx within B
    int idyB = idt / DIM_XB;    // idy within B

    int blx = blockIdx.x;   // block's m dimension
    int bly = blockIdx.y;   // block's n dimension

    // Registers for the innermost loop
    T rC[THR_N][THR_M];
    T rA[THR_M];
    T rB[THR_N];

    // Registers for the dev->shmem copy
    T ra[BLK_M/DIM_YA][BLK_K/DIM_XA];
    T rb[BLK_N/DIM_YB][BLK_K/DIM_XB];

    // bound is the correction to offs_d in order to not get out of memory bound
    // so bound could be negative value since offs_d could be out of bound
    T *offs_dA = A + blx*BLK_M*LDA + idyA*LDA + idxA;
    int boundA = (LDA*(M-1) + K) - ( blx*BLK_M*LDA + idyA*LDA + idxA ) -1;

    T *offs_dB = B + bly*BLK_N*LDB + idyB*LDB + idxB;
    int boundB = (LDB*(N-1) + K) - ( bly*BLK_N*LDB + idyB*LDB + idxB ) -1;

    int m, n, k, kk;

    // Zero C
    #pragma unroll
    for (n = 0; n < THR_N; n++)
        #pragma unroll
        for (m = 0; m < THR_M; m++)
            rC[n][m] = 0.0;

    // Load A dev->shmem
    #pragma unroll
    for (n = 0; n < BLK_M; n += DIM_YA)
        #pragma unroll
        for (m = 0; m < BLK_K; m += DIM_XA)
            sA(n+idyA, m+idxA) = fetch(A, m, n, boundA);

    #pragma unroll
    for (n = 0; n < BLK_N; n += DIM_YB)
        #pragma unroll
        for (m = 0; m < BLK_K; m += DIM_XB)
            sB(m+idxB, n+idyB) = fetch(B, m, n, boundB);

    __syncthreads();

    for (kk = 0; kk < K-BLK_K; kk += BLK_K) {
        offs_dA += BLK_K;
        boundA  -= BLK_K;

        offs_dB += BLK_K;
        boundB  -= BLK_K;

        // Load A dev->regs
        #pragma unroll
        for (n = 0; n < BLK_M/DIM_YA; n++)
            #pragma unroll
            for (m = 0; m < BLK_K/DIM_XA; m++)
                ra[n][m] = fetch(A, m*DIM_XA, n*DIM_YA, boundA);

        // Load B dev->regs
        #pragma unroll
        for (n = 0; n < BLK_N/DIM_YB; n++)
            #pragma unroll
            for (m = 0; m < BLK_K/DIM_XB; m++)
                rb[n][m] = fetch(B, m*DIM_XB, n*DIM_YB, boundB);

        // Multiply
        #pragma unroll
        for (k = 0; k < BLK_K; k++) {
            // Load A shmem->regs
            #pragma unroll
            for (m = 0; m < THR_M; m++)
                rA[m] = sA(m*DIM_X+idx, k);

            // Load B shmem->regs
            #pragma unroll
            for (n = 0; n < THR_N; n++)
                rB[n] = sB(k, n*DIM_Y+idy);

            // Compute
            #pragma unroll
            for (n = 0; n < THR_N; n++) {
                #pragma unroll
                for (m = 0; m < THR_M; m++) {
                    rC[n][m] += rA[m] * rB[n];
                }
            }
        }

        __syncthreads();

        // Load A regs->shmem
        #pragma unroll
        for (n = 0; n < BLK_M/DIM_YA; n++)
            #pragma unroll
            for (m = 0; m < BLK_K/DIM_XA; m++)
                sA(n*DIM_YA+idyA, m*DIM_XA+idxA) = ra[n][m];

        // Load B regs->shmem
        #pragma unroll
        for (n = 0; n < BLK_N/DIM_YB; n++)
            #pragma unroll
            for (m = 0; m < BLK_K/DIM_XB; m++)
                sB(m*DIM_XB+idxB, n*DIM_YB+idyB) = rb[n][m];

        __syncthreads();
    }

    // Multiply last full (BLK_K) or partial block of
    // columns of op(A) and rows of op(B).
    // It's okay that m,n exceed matrix bounds as all work is in registers
    // or shared memory, and out-of-bounds rC[n][m] will not be saved later.
    kk = K - kk;
    #pragma unroll
    for (k = 0; k < kk; k++)
    {
        // Load A shmem->regs
        #pragma unroll
        for (m = 0; m < THR_M; m++)
            rA[m] = sA(m*DIM_X+idx, k);

        // Load B shmem->regs
        #pragma unroll
        for (n = 0; n < THR_N; n++)
            rB[n] = sB(k, n*DIM_Y+idy);

        // Compute
        #pragma unroll
        for (n = 0; n < THR_N; n++) {
            #pragma unroll
            for (m = 0; m < THR_M; m++) {
                rC[n][m] += rA[m] * rB[n];
            }
        }
    }

    // Store C regs->dev
    #pragma unroll
    for (n = 0; n < THR_N; n++) {
        int coord_dCn = bly*BLK_N + n*DIM_Y + idy;
        #pragma unroll
        for (m = 0; m < THR_M; m++) {
            int coord_dCm = blx*BLK_M + m*DIM_X + idx;
            if (coord_dCm < M && coord_dCn < N) {
                int offsC = coord_dCn*LDC + coord_dCm;
                atomicAdd(C + offsC, rC[n][m]);
            }
        }
    }
}


/******************************************************************************/
template <typename T, int DIM_X, int DIM_Y,
         int BLK_M, int BLK_N, int BLK_K,
         int DIM_XA, int DIM_YA,
         int DIM_XB, int DIM_YB>
static __global__
void vbatched_gemm_kernel(
    int* M, int* N, int K,
    T * * global_A_array, int* global_lda,
    T * * global_B_array, int* global_ldb,
    T              ** global_C_array, int* global_ldc)
{
    extern __shared__ __align__(sizeof(T)) unsigned char smem[];
    T *shared_mem = reinterpret_cast<T *>(smem);

    int batchid = blockIdx.z;
    int local_M = (int)M[batchid];
    int local_N = (int)N[batchid];

    if( blockIdx.x >= (local_M+BLK_M-1)/BLK_M ) return;
    if( blockIdx.y >= (local_N+BLK_N-1)/BLK_N ) return;

    int shared_lda = BLK_M+1;
    int shared_ldb = BLK_K+1;
    T* shared_A = (T*)shared_mem;
    T* shared_B = shared_A + shared_lda * BLK_K;

    vbatched_gemm_device<T, DIM_X, DIM_Y, 
                         BLK_M, BLK_N, BLK_K,
                         DIM_XA, DIM_YA,
                         DIM_XB, DIM_YB, 
                         (BLK_M/DIM_X), (BLK_N/DIM_Y)>
                        (local_M, local_N, K,
                        global_A_array[batchid], (int)global_lda[batchid],
                        global_B_array[batchid], (int)global_ldb[batchid],
                        global_C_array[batchid], (int)global_ldc[batchid],
                        shared_A, shared_lda, shared_B, shared_ldb);
}

static inline int ceildiv( int x, int y )
{
    return (x + y - 1)/y;
}

template <typename T, int DIM_X, int DIM_Y,
         int BLK_M, int BLK_N, int BLK_K,
         int DIM_XA, int DIM_YA,
         int DIM_XB, int DIM_YB>
void vbatched_gemm_impl(int max_m, int max_n,
                 int* m, int* n, int k,
                 T  * * global_A_array, int* global_lda,
                 T * * global_B_array, int* global_ldb,
                 T ** global_C_array, int* global_ldc,
                 int batchCount, hipStream_t stream)
{
    // The positions of A and B have been swapped here.
    // This is because the original code is for column-major matrices.
    // We use row-major matrices, so we need to swap A and B.
    // The vbatched_gemm_impl is for C = trans(A) * B + C, but we need trans(C).
    // Which means: trans(C) = trans(trans(A)*B + C) = trans(B) * A + trans(C)
    // Then, ldc should be N, lda and ldb should be K

    size_t shared_mem_size = 0;
    shared_mem_size += (BLK_M+1) * BLK_K * sizeof(T);
    shared_mem_size += (BLK_K+1) * BLK_N * sizeof(T);
    dim3 dimBlock(DIM_X, DIM_Y);
    const int max_batch_count = 32768;
    const int loop_num = batchCount / max_batch_count;
    const int remain_num = batchCount % max_batch_count;
    for (int i = 0; i < loop_num; ++i)
    {
        dim3 dimGrid(ceildiv( max_n, BLK_M ), ceildiv( max_m, BLK_N ), max_batch_count);
        vbatched_gemm_kernel<T, DIM_X, DIM_Y,
                         BLK_M, BLK_N, BLK_K,
                         DIM_XA, DIM_YA,
                         DIM_XB, DIM_YB>
                         <<<dimGrid, dimBlock, shared_mem_size, stream>>>
                         (n + i * max_batch_count, m + i * max_batch_count, k,
                         global_B_array + i * max_batch_count, global_ldb + i * max_batch_count,
                         global_A_array + i * max_batch_count, global_lda + i * max_batch_count,
                         global_C_array + i * max_batch_count, global_ldc + i * max_batch_count);
        checkCudaLastError();
    }
    if (remain_num > 0)
    {
        dim3 dimGrid(ceildiv( max_n, BLK_M ), ceildiv( max_m, BLK_N ), remain_num);
        vbatched_gemm_kernel<T, DIM_X, DIM_Y,
                         BLK_M, BLK_N, BLK_K,
                         DIM_XA, DIM_YA,
                         DIM_XB, DIM_YB>
                         <<<dimGrid, dimBlock, shared_mem_size, stream>>>
                         (n + loop_num * max_batch_count, m + loop_num * max_batch_count, k,
                         global_B_array + loop_num * max_batch_count, global_ldb + loop_num * max_batch_count,
                         global_A_array + loop_num * max_batch_count, global_lda + loop_num * max_batch_count,
                         global_C_array + loop_num * max_batch_count, global_ldc + loop_num * max_batch_count);    
        checkCudaLastError();
    }
}

template <typename T, int DIM_X, int DIM_Y, int BLK_M, int BLK_N, int BLK_K,
          int DIM_XA, int DIM_YA, int DIM_XB, int DIM_YB>
void gemm_time_measure(int max_m, int max_n,
                 int* m, int* n, int k,
                 T ** global_A_array, int* global_lda,
                 T ** global_B_array, int* global_ldb,
                 T ** global_C_array, int* global_ldc,
                 int batchCount, hipStream_t stream, float &fast_time, func_type &fastest_algo,
                 double *cpu_result, double * h_global_C, double *d_global_C)
{
    hipEvent_t start, stop;
    checkCuda(hipMemset(d_global_C, 0, batchCount * max_m * max_n * sizeof(double)));
    checkCuda(hipEventCreate(&start));
    checkCuda(hipEventCreate(&stop));
    checkCuda(hipEventRecord(start, stream));
    vbatched_gemm_impl<T, DIM_X, DIM_Y, BLK_M, BLK_N, BLK_K, DIM_XA, DIM_YA, DIM_XB, DIM_YB>
                    (max_m, max_n, m, n, k,
                    global_A_array, global_lda,
                    global_B_array, global_ldb,
                    global_C_array, global_ldc,
                    batchCount, stream);
    checkCuda(hipEventRecord(stop, stream));
    hipError_t cuda_status = hipGetLastError();
    checkCuda(hipStreamSynchronize(stream));
    float milliseconds = 0;
    checkCuda(hipEventElapsedTime(&milliseconds, start, stop));

    // WARNING !!!!! Here we assume that all m and n are the same
    checkCuda(hipMemcpy(h_global_C, d_global_C, batchCount * max_m * max_n * sizeof(double), hipMemcpyDeviceToHost));  
    bool check_result = true;
    for (int i = 0; i < batchCount * max_m * max_n; ++i)
    {
        if (abs(cpu_result[i] - h_global_C[i]) > 0.001)
        {
            check_result = false;
            break;
        }
    }
    if (milliseconds < fast_time && cuda_status == hipSuccess && check_result)
    {
        fast_time = milliseconds;
        fastest_algo = vbatched_gemm_impl<T, DIM_X, DIM_Y, BLK_M, BLK_N, BLK_K, DIM_XA, DIM_YA, DIM_XB, DIM_YB>;
        #ifdef __DEBUG
        std::cout << "found! fastest time: " << fast_time << std::endl;
        std::cout << DIM_X << ","<< DIM_Y<< ","<< BLK_M<< ","<< BLK_N<< ","<< BLK_K<< ","<< DIM_XA<< ","<< DIM_YA<< ","<< DIM_XB<< ","<< DIM_YB << std::endl;
        #endif
    }
}

void gemm_algo_selector(int matrix_k, func_type & fastest_algo)
{

    int batchCount_per_type = 32;
    int batchCount = batchCount_per_type * GlobalC::ucell.ntype * GlobalC::ucell.ntype;
    int *h_m = new int[batchCount];
    int *h_n = new int[batchCount];
    int *h_global_lda = new int[batchCount];
    int *h_global_ldb = new int[batchCount];
    int *h_global_ldc = new int[batchCount];
    int max_m = GlobalC::ucell.nwmax, max_n = GlobalC::ucell.nwmax;
    double **h_global_A_array = new double *[batchCount];
    double **h_global_B_array = new double *[batchCount];
    double **h_global_C_array = new double *[batchCount];

    double *h_global_A = new double[batchCount * max_m * matrix_k];
    double *h_global_B = new double[batchCount * max_n * matrix_k];
    double *h_global_C = new double[batchCount * max_m * max_n];

    for (int i = 0; i < batchCount * max_m * matrix_k; ++i)
    {
        h_global_A[i] = i * 0.001;
    }
    for (int i = 0; i < batchCount * max_n * matrix_k; ++i)
    {
        h_global_B[i] = i * 0.002;
    }
    memset(h_global_C, 0, batchCount * max_m * max_n * sizeof(double));

    // Allocate device memory
    int *d_m;
    int *d_n;
    int *d_global_lda;
    int *d_global_ldb;
    int *d_global_ldc;

    double **d_global_A_array;
    double **d_global_B_array;
    double **d_global_C_array;

    double *d_global_A;
    double *d_global_B;
    double *d_global_C;

    double *cpu_result = new double[batchCount * max_m * max_n];
    memset(cpu_result, 0, batchCount * max_m * max_n * sizeof(double));

    checkCuda(hipMalloc(&d_m, batchCount * sizeof(int)));
    checkCuda(hipMalloc(&d_n, batchCount * sizeof(int)));
    checkCuda(hipMalloc(&d_global_lda, batchCount * sizeof(int)));
    checkCuda(hipMalloc(&d_global_ldb, batchCount * sizeof(int)));
    checkCuda(hipMalloc(&d_global_ldc, batchCount * sizeof(int)));
    checkCuda(hipMalloc(&d_global_A_array, batchCount * sizeof(double *)));
    checkCuda(hipMalloc(&d_global_B_array, batchCount * sizeof(double *)));
    checkCuda(hipMalloc(&d_global_C_array, batchCount * sizeof(double *)));

    checkCuda(hipMalloc(&d_global_A, batchCount * max_m * matrix_k * sizeof(double)));
    checkCuda(hipMalloc(&d_global_B, batchCount * max_n * matrix_k * sizeof(double)));
    checkCuda(hipMalloc(&d_global_C, batchCount * max_m * max_n * sizeof(double)));
     
    checkCuda(hipMemset(d_global_C, 0, batchCount * max_m * max_n * sizeof(double)));
    int index = 0;
    for (int i = 0; i < batchCount_per_type; ++i)
    {
        for (int j = 0; j < GlobalC::ucell.ntype; j++)
        {
            for (int k = 0; k < GlobalC::ucell.ntype; k++)
            {
                h_m[index] = GlobalC::ucell.atoms[j].nw;
                h_n[index] = GlobalC::ucell.atoms[k].nw;
                h_global_lda[index] = matrix_k;
                h_global_ldb[index] = matrix_k;
                h_global_ldc[index] = GlobalC::ucell.atoms[k].nw;

                h_global_A_array[index] = &d_global_A[index * max_m * matrix_k];
                h_global_B_array[index] = &d_global_B[index * max_n * matrix_k];
                h_global_C_array[index] = &d_global_C[index * max_n * max_m]; // test atom add
                BlasConnector::gemm('N', 'T', h_m[index], h_n[index], matrix_k, 1.0, &h_global_A[index * max_m * matrix_k], matrix_k, &h_global_B[index * max_n * matrix_k], matrix_k, 1.0, &cpu_result[index * max_m * max_n], h_n[index]);
                index++;
            }
        }
    }

    checkCuda(hipMemcpy(d_m, h_m, batchCount * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_n, h_n, batchCount * sizeof(int), hipMemcpyHostToDevice));

    checkCuda(hipMemcpy(d_global_lda, h_global_lda, batchCount * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_global_ldb, h_global_ldb, batchCount * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_global_ldc, h_global_ldc, batchCount * sizeof(int), hipMemcpyHostToDevice));

    checkCuda(hipMemcpy(d_global_A_array, h_global_A_array, batchCount * sizeof(double *), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_global_B_array, h_global_B_array, batchCount * sizeof(double *), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_global_C_array, h_global_C_array, batchCount * sizeof(double *), hipMemcpyHostToDevice));

    checkCuda(hipMemcpy(d_global_A, h_global_A, batchCount * max_m * matrix_k * sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_global_B, h_global_B, batchCount * max_n * matrix_k * sizeof(double), hipMemcpyHostToDevice));

    hipStream_t temp_stream;
    checkCuda(hipStreamCreate(&temp_stream));

    float fastest_time = 1000000;
    fastest_algo = vbatched_gemm_impl<double, 16, 4, 32, 16, 16, 16, 4, 16, 4>;
    #include"code_gen.cpp"
    checkCuda(hipStreamDestroy(temp_stream));
    std::cout << " gemm_algo_selector::Fastest time: " << fastest_time << " ms" << std::endl;
    // fastest_algo = vbatched_gemm_impl<double, 16, 4, 32, 16, 16, 16, 4, 16, 4>;
    delete[] h_global_A_array;
    delete[] h_global_B_array;
    delete[] h_global_C_array;

    delete[] h_m;
    delete[] h_n;

    delete[] h_global_lda;
    delete[] h_global_ldb;
    delete[] h_global_ldc;

    delete[] h_global_A;
    delete[] h_global_B;
    delete[] h_global_C;

    delete[] cpu_result;

    // Cleanup
    checkCuda(hipFree(d_global_A_array));
    checkCuda(hipFree(d_global_B_array));
    checkCuda(hipFree(d_global_C_array));

    checkCuda(hipFree(d_m));
    checkCuda(hipFree(d_n));

    checkCuda(hipFree(d_global_lda));
    checkCuda(hipFree(d_global_ldb));
    checkCuda(hipFree(d_global_ldc));


    checkCuda(hipFree(d_global_A));
    checkCuda(hipFree(d_global_B));
    checkCuda(hipFree(d_global_C));

}